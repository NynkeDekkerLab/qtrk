#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "std_incl.h"
#include "utils.h"

#include <cassert>
#include <cstdlib>
#include <stdio.h>
#include <windows.h>
#include <cstdarg>
#include <valarray>

#include "random_distr.h"

#include <stdint.h>
#include "gpu_utils.h"
#include "QueuedCUDATracker.h"

#include "simplefft.h"
#include "cudafft/cudafft.h"

double getPreciseTime()
{
	uint64_t freq, time;

	QueryPerformanceCounter((LARGE_INTEGER*)&time);
	QueryPerformanceFrequency((LARGE_INTEGER*)&freq);

	return (double)time / (double)freq;
}


std::string getPath(const char *file)
{
	std::string s = file;
	int pos = s.length()-1;
	while (pos>0 && s[pos]!='\\' && s[pos]!= '/' )
		pos--;
	
	return s.substr(0, pos);
}


inline __device__ float2 mul_conjugate(float2 a, float2 b)
{
	float2 r;
	r.x = a.x*b.x + a.y*b.y;
	r.y = a.y*b.x - a.x*b.y;
	return r;
}

texture<float, hipTextureType2D, hipReadModeElementType> smpImgRef(0, hipFilterModeLinear);


void TestSimpleFFT()
{
	int N=64;
	cudafft<double> fft(N, false);

	std::vector< cudafft<double>::cpx_type > data(N), result(N), cpu_result(N);
	for (int x=0;x<N;x++) {
		data[x].x = 10*cos(x*0.1f-5);
		data[x].y = 6*cos(x*0.2f-2)+3;
	}

	std::vector<sfft::complex<double> > twiddles = sfft::fill_twiddles<double>(N);

	fft.host_transform(&data[0], &cpu_result[0]);
	sfft::fft_forward(N, (sfft::complex<double>*)&data[0], &twiddles[0]);
	
	for (int k=0;k<N;k++) {
		dbgprintf("[%d] kissfft: %f+%fi, sfft: %f+%fi. diff=%f+%fi\n", k, cpu_result[k].x, cpu_result[k].y, data[k].x,data[k].y, cpu_result[k].x - data[k].x,cpu_result[k].y - data[k].y);
	}
}


void ShowCUDAError() {
	hipError_t err = hipGetLastError();
	dbgprintf("Cuda error: %s\n", hipGetErrorString(err));
}

__global__ void testJobPassing(CUDATrackerJob job, CUDATrackerJob* a)
{
	CUDATrackerJob* a0 = &a[0];
	CUDATrackerJob* a1 = &a[1];
}

void TestJobPassing()
{
	CUDATrackerJob job;
	job.id = 1;
	job.initialPos.x = 2;
	job.initialPos.y = 3;
	job.initialPos.z = 4;
	job.zlut = 5;
	job.zlutPlane = 6;
	job.locType = LocalizeBuildZLUT;

	std::vector<CUDATrackerJob> jobs;
	jobs.push_back(job);
	jobs.push_back(job);

	testJobPassing<<<dim3(),dim3()>>>(job, device_vec<CUDATrackerJob> (jobs).data);
}

void TestLocalization()
{
#ifdef _DEBUG
	const int NumImages=4;
#else
	const int NumImages=256;
#endif
	int N = 10;
	QTrkSettings cfg;
	cfg.numThreads = -1;
	cfg.qi_iterations = 2;
	cfg.qi_maxradius = 30;
	QueuedCUDATracker trk(&cfg);

	auto images = cudaImageListf::alloc(80,80, NumImages, trk.UseHostEmulate());
	ShowCUDAError();
	std::vector<float3> positions(images.count);
	{
	device_vec< sfft::complex<float> > test;
	test = trk.DeviceMem( std::vector < sfft::complex<float> > (3) );
	}
	for(int i=0;i<images.count;i++) {
		float xp = images.w/2+(rand_uniform<float>() - 0.5) * 5;
		float yp = images.h/2+(rand_uniform<float>() - 0.5) * 5;
		positions[i] = make_float3(xp, yp, 3);
	}
	device_vec<float3> d_pos = trk.DeviceMem(positions);

	dbgprintf("Generating... %d images\n", N*images.count);

	double t0 = getPreciseTime();
	for (int i=0;i<N;i++) 
		trk.GenerateImages(images, d_pos.data);
	hipDeviceSynchronize();
	double tgen = getPreciseTime() - t0;

	auto d_com = trk.DeviceMem<float2>(positions.size());
	auto d_qi = trk.DeviceMem<float2>(positions.size());
	double t1 = getPreciseTime();
	dbgprintf("COM\n");
	for (int i=0;i<N;i++)
		trk.ComputeBgCorrectedCOM(images, d_com.data);
	hipDeviceSynchronize();
	double t2 = getPreciseTime();
	double tcom = t2 - t1;

	dbgprintf("QI\n");
	for (int i=0;i<N;i++)
		trk.ComputeQI(images, d_com.data, d_qi.data);
	hipDeviceSynchronize();
	double tqi = getPreciseTime() - t2;

	std::vector<float2> com(d_com), qi(d_qi);
	/*
	for (int i=0;i<images.count;i++) {
		dbgprintf("[%d] true pos=( %.4f, %.4f ).  COM error=( %.4f, %.4f ).  QI error=( %.4f, %.4f ) \n", i, 
			positions[i].x, positions[i].y, com[i].x - positions[i].x, com[i].y - positions[i].y, qi[i].x - positions[i].x, qi[i].y - positions[i].y );
	}*/

	N *= images.count;
	dbgprintf("Image generating: %f img/s. COM: %f img/s. QI: %f img/s\n", N/tgen, N/tcom, N/tqi);

	ShowCUDAError();
	images.free();
}



void QTrkTest()
{
	QTrkSettings cfg;
	cfg.width = cfg.height = 128;
	cfg.qi_iterations = 3;
	cfg.qi_maxradius = 50;
	cfg.xc1_iterations = 2;
	cfg.xc1_profileLength = 64;
	cfg.numThreads = 0; // direct processing, dont use queue
	//cfg.numThreads = 6;
	QueuedCUDATracker qtrk(&cfg);
	float *image = new float[cfg.width*cfg.height];

	// Generate ZLUT
	int radialSteps=64, zplanes=100;
	float zmin=0.5,zmax=3;
	qtrk.SetZLUT(0, 1, zplanes, radialSteps);
	qtrk.Start();
	for (int x=0;x<zplanes;x++)  {
		vector2f center = { cfg.width/2, cfg.height/2 };
		float s = zmin + (zmax-zmin) * x/(float)(zplanes-1);
		GenerateTestImage(ImageData(image, cfg.width, cfg.height), center.x, center.y, s, 0.0f);
		qtrk.ScheduleLocalization((uchar*)image, cfg.width*sizeof(float),QTrkFloat, (LocalizeType)(LocalizeBuildZLUT|LocalizeQI), x, 0, 0, x);
	}
	qtrk.Flush();
	// wait to finish ZLUT
	while(true) {
		int rc = qtrk.GetResultCount();
		if (rc == zplanes) break;
		Sleep(100);
		dbgprintf(".");
	}
	float* zlut = qtrk.GetZLUT(0,0,0);
	qtrk.ClearResults();
	uchar* zlut_bytes = floatToNormalizedInt(zlut, radialSteps, zplanes, (uchar)255);
	WriteJPEGFile(zlut_bytes, radialSteps, zplanes, "qtrkzlutcuda.jpg", 99);
	delete[] zlut; delete[] zlut_bytes;
	
	// Schedule images to localize on
	int NumImages=10, JobsPerImg=2000;
	dbgprintf("Generating %d images...\n", NumImages);
	double tgen = 0.0, tschedule = 0.0;
	std::vector<float> truepos(NumImages*3);
	for (int n=0;n<NumImages;n++) {
		double t1 = getPreciseTime();
		float xp = cfg.width/2+(rand_uniform<float>() - 0.5) * 5;
		float yp = cfg.height/2+(rand_uniform<float>() - 0.5) * 5;
		float z = zmin + 0.1f + (zmax-zmin-0.2f) * rand_uniform<float>();
		truepos[n*3+0] = xp;
		truepos[n*3+1] = yp;
		truepos[n*3+2] = z;

		GenerateTestImage(ImageData(image, cfg.width, cfg.height), xp, yp, z, 10000);
		double t2 = getPreciseTime();
		for (int k=0;k<JobsPerImg;k++)
			qtrk.ScheduleLocalization((uchar*)image, cfg.width*sizeof(float), QTrkFloat, (LocalizeType)(LocalizeQI|LocalizeZ), n, 0, 0, 0);
		double t3 = getPreciseTime();
		tgen += t2-t1;
		tschedule += t3-t2;
	}
	delete[] image;
	dbgprintf("Schedule time: %f, Generation time: %f\n", tschedule, tgen);

	// Measure speed
	dbgprintf("Localizing on %d images...\n", NumImages*JobsPerImg);
	double tstart = getPreciseTime();
	int total = NumImages*JobsPerImg;
	qtrk.Flush();
	int rc = 0, displayrc=0;
	do {
		rc = qtrk.GetResultCount();
		while (displayrc<rc) {
			if( displayrc%JobsPerImg==0) dbgprintf("Done: %d / %d\n", displayrc, total);
			displayrc++;
		}
		Sleep(10);
	} while (rc != total);
	double tend = getPreciseTime();

	// Wait for last jobs
	double errX=0.0, errY=0.0, errZ=0.0;

	for (int i=0;i<total;i++) {
		LocalizationResult result;

		if (qtrk.PollFinished(&result, 1)) {
			int iid = result.id;
			float x = fabs(truepos[iid*3+0]-result.pos.x);
			float y = fabs(truepos[iid*3+1]-result.pos.y);
			result.z = zmin + (zmax-zmin) * result.z / (float)(zplanes-1); // transform from index scale to coordinate scale
			float z = fabs(truepos[iid*3+2]-result.z);
		//	dbgprintf("ID: %d. Boundary Error:%d. ErrX=%f, ErrY=%f, ErrZ=%f\n", result.id, result.error, x,y,z);
			errX += x; errY += y; errZ += z;
		}
	}
	dbgprintf("Localization Speed: %d (img/s)\n", (int)( total/(tend-tstart) ));
	dbgprintf("ErrX: %f, ErrY: %f, ErrZ: %f\n", errX/total, errY/total,errZ/total);
}


int main(int argc, char *argv[])
{
//	testLinearArray();

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);

	std::string path = getPath(argv[0]);

	//TestJobPassing();
	//TestLocalization();
	//TestSimpleFFT();
	//TestKernelFFT();
	QTrkTest();

	return 0;
}
