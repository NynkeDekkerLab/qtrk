#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "std_incl.h"
#include "utils.h"

#include <cassert>
#include <cstdlib>
#include <stdio.h>
#include <windows.h>
#include <cstdarg>
#include <valarray>

#include "cudafft/cudafft.h"
#include "random_distr.h"

#include <stdint.h>
#include "cudaImageList.h"
#include "QueuedCUDATracker.h"

double getPreciseTime()
{
	uint64_t freq, time;

	QueryPerformanceCounter((LARGE_INTEGER*)&time);
	QueryPerformanceFrequency((LARGE_INTEGER*)&freq);

	return (double)time / (double)freq;
}





std::string getPath(const char *file)
{
	std::string s = file;
	int pos = s.length()-1;
	while (pos>0 && s[pos]!='\\' && s[pos]!= '/' )
		pos--;
	
	return s.substr(0, pos);
}

void TestLocalization()
{
	int repeat = 10;
	int xcorProfileLen = 128, xcorProfileWidth = 16;
	float t_gen=0, t_com=0, t_xcor=0;

	hipEvent_t gen_start, gen_end, com_start, com_end, xcor_end;
	hipEventCreate(&gen_start);
	hipEventCreate(&gen_end);
	hipEventCreate(&com_start);
	hipEventCreate(&com_end);
	hipEventCreate(&xcor_end);

	// Create some space for images
	cudaImageList images = cudaImageList::alloc(170,150, 2048);
	dbgprintf("Image memory used: %d bytes\n", images.totalsize());
	float3* d_pos;
	hipMalloc(&d_pos, sizeof(float3)*images.count);
	float2* d_com;
	hipMalloc(&d_com, sizeof(float2)*images.count);
	float2* d_xcor;
	hipMalloc(&d_xcor, sizeof(float2)*images.count);

	float3* positions = new float3[images.count];
	for(int i=0;i<images.count;i++) {
		float xp = images.w/2+(rand_uniform<float>() - 0.5) * 5;
		float yp = images.h/2+(rand_uniform<float>() - 0.5) * 5;
		positions[i] = make_float3(xp, yp, 10);
	}
	hipMemcpy(d_pos, positions, sizeof(float3)*images.count, hipMemcpyHostToDevice);
	double comErr=0.0, xcorErr=0.0;
	QTrkSettings cfg;
	QueuedCUDATracker qtrk(&cfg);

	for (int k=0;k<repeat;k++) {
		hipEventRecord(gen_start);
		qtrk.GenerateImages(images, d_pos);
		hipEventRecord(gen_end);

		hipEventRecord(com_start);
		qtrk.ComputeBgCorrectedCOM(images, d_com);
		hipEventRecord(com_end);
		hipEventSynchronize(com_end);

		float t_gen0, t_com0, t_xcor0;
		hipEventElapsedTime(&t_gen0, gen_start, gen_end);
		t_gen+=t_gen0;
		hipEventElapsedTime(&t_com0, com_start, com_end);
		t_com+=t_com0;
		std::vector<float2> com(images.count);
		hipMemcpyAsync(&com[0], d_com, sizeof(float2)*images.count, hipMemcpyDeviceToHost);

		qtrk.Compute1DXCor(images, d_com, d_xcor);
		hipEventRecord(xcor_end);
		hipEventSynchronize(xcor_end);
		hipEventElapsedTime(&t_xcor0, com_end, xcor_end);
		t_xcor+=t_xcor0;

		std::vector<float2> xcor(images.count);
		hipMemcpy(&xcor[0], d_xcor, sizeof(float2)*images.count, hipMemcpyDeviceToHost);

		float dx,dy;
		for (int i=0;i<images.count;i++) {
			dx = (com[i].x-positions[i].x);
			dy = (com[i].y-positions[i].y);
			comErr += sqrt(dx*dx+dy*dy);

			dx = (xcor[i].x-positions[i].x);
			dy = (xcor[i].y-positions[i].y);
			xcorErr += sqrt(dx*dx+dy*dy);
		}
	}


	int N = images.count*repeat*1000; // times are in ms
	dbgprintf("COM error: %f pixels. XCor error: %f pixels\n",comErr/(images.count*repeat), xcorErr/(images.count*repeat));
	dbgprintf("Image generating: %f img/s. COM computation: %f img/s. 1D XCor: %f img/s\n", N/t_gen, N/t_com, N/t_xcor);
	hipFree(d_com);
	hipFree(d_pos);
	images.free();

	hipEventDestroy(gen_start); hipEventDestroy(gen_end); 
	hipEventDestroy(com_start); hipEventDestroy(com_end); 
}

__global__ void runCudaFFT(cudafft<float>::cpx_type *src, cudafft<float>::cpx_type *dst, cudafft<float>::KernelParams kparams)
{
	cudafft<float>::transform(src,dst, kparams);
}

void TestKernelFFT()
{
	int N=64;
	cudafft<float> fft(N, false);

	std::vector< cudafft<float>::cpx_type > data(N), result(N);
	for (int x=0;x<N;x++)
		data[x].x = 10*cos(x*0.1f);

	fft.host_transform(&data[0], &result[0]);
	for (int x=0;x<N;x++)
		dbgprintf("[%d] %f+%fi\n", x, result[x].x, result[x].y);

	// now put data in video mem
	cudafft<float>::cpx_type *src,*d_result;
	hipMalloc(&src, sizeof(cudafft<float>::cpx_type)*N);
	hipMemcpy(src, &data[0], sizeof(cudafft<float>::cpx_type)*N, hipMemcpyHostToDevice);
	hipMalloc(&d_result, sizeof(cudafft<float>::cpx_type)*N);

	runCudaFFT<<<dim3(1),dim3(1)>>>(src,d_result, fft.kparams);

	std::vector< cudafft<float>::cpx_type > result2(N);
	hipMemcpy(&result2[0], d_result, sizeof(cudafft<float>::cpx_type)*N, hipMemcpyDeviceToHost);

	for (int i=0;i<N;i++) {
		cudafft<float>::cpx_type d=result2[i]-result[i];
		dbgprintf("[%d] %f+%fi\n", i, d.x, d.y);
	}

	hipFree(src);
	hipFree(d_result);

}

int main(int argc, char *argv[])
{
//	testLinearArray();

	std::string path = getPath(argv[0]);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

//	TestLocalization();

	TestKernelFFT();

	
	return 0;
}
