#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "std_incl.h"
#include "utils.h"

#include <cassert>
#include <cstdlib>
#include <stdio.h>
#include <windows.h>
#include <cstdarg>
#include <valarray>

#include "random_distr.h"

#include <stdint.h>
#include "gpu_utils.h"
#include "QueuedCUDATracker.h"
#include "queued_cpu_tracker.h"



std::string getPath(const char *file)
{
	std::string s = file;
	int pos = s.length()-1;
	while (pos>0 && s[pos]!='\\' && s[pos]!= '/' )
		pos--;
	
	return s.substr(0, pos);
}


inline __device__ float2 mul_conjugate(float2 a, float2 b)
{
	float2 r;
	r.x = a.x*b.x + a.y*b.y;
	r.y = a.y*b.x - a.x*b.y;
	return r;
}


void ShowCUDAError() {
	hipError_t err = hipGetLastError();
	dbgprintf("Cuda error: %s\n", hipGetErrorString(err));
}

__shared__ float cudaSharedMem[];

__device__ float compute(int idx, float* buf, int s)
{
	// some random calcs to make the kernel unempty
	float k=0.0f;
	for (int x=0;x<s;x++ ){
		k+=cosf(x*0.1f*idx);
		buf[x]=k;
	}
	for (int x=0;x<s/2;x++){
		buf[x]=buf[x]*buf[x];
	}
	float sum=0.0f;
	for (int x=s-1;x>=1;x--) {
		sum += buf[x-1]/(fabsf(buf[x])+0.1f);
	}
	return sum;
}


__global__ void testWithGlobal(int n, int s, float* result, float* buf) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		result [idx] = compute(idx, &buf [idx * s],s);
	}
}

__global__ void testWithShared(int n, int s, float* result) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		result [idx] = compute(idx, &cudaSharedMem[threadIdx.x * s],s);
	}
}

void TestSharedMem()
{
	int n=100, s=200;
	dim3 nthreads(32), nblocks( (n+nthreads.x-1)/nthreads.x);
	device_vec<float> buf(n*s);
	device_vec<float> result_s(n), result_g(n);

	double t0 = GetPreciseTime();
	testWithGlobal<<<nblocks,nthreads>>>(n,s,result_g.data,buf.data);
	hipDeviceSynchronize();
	double t1 = GetPreciseTime();
	testWithShared <<<nblocks,nthreads,s*sizeof(float)*nthreads.x>>>(n,s,result_s.data);
	hipDeviceSynchronize();
	double t2 = GetPreciseTime();

	std::vector<float> rs = result_s, rg = result_g;
	for (int x=0;x<n;x++) {
		dbgprintf("result_s[%d]=%f.   result_g[%d]=%f\n", x,rs[x], x,rg[x]);
	}

	dbgprintf("Speed of shared comp: %f, speed of global comp: %f\n", n/(t2-t1), n/(t1-t0));
}

void FloatToJPEGFile (const char *name, float* d, int w,int h)
{
	uchar* zlut_bytes = floatToNormalizedInt(d, w,h, (uchar)255);
	WriteJPEGFile(zlut_bytes, w, h, name, 99);
	delete[] zlut_bytes;
}

void QTrkTest()
{
	QTrkSettings cfg;
	cfg.width = cfg.height = 120;
	cfg.qi_iterations = 1;
	cfg.qi_maxradius = 25;
	cfg.xc1_iterations = 2;
	cfg.xc1_profileLength = 64;
	cfg.numThreads = -1;
	cfg.com_bgcorrection = 0.0f;
	cfg.zlut_maxradius = 30;
	cfg.zlut_radialsteps = 64;
	cfg.zlut_angularsteps = 128;
	bool haveZLUT = false;
#ifdef _DEBUG
	cfg.qi_radialsteps=16;
	cfg.numThreads = 2;
	cfg.qi_iterations=1;
	int total= 10;
	int batchSize = 2;
	haveZLUT=false;
#else
	cfg.numThreads = 4;
	int total = 30000;
	int batchSize = 512;
#endif

	QueuedCUDATracker qtrk(&cfg, batchSize);
	QueuedCPUTracker qtrkcpu(&cfg);
	float *image = new float[cfg.width*cfg.height];
	bool cpucmp = true;

	qtrk.EnableTextureCache(true);

	srand(1);

	// Generate ZLUT
	int zplanes=100;
	float zmin=0.5,zmax=3;
	qtrk.SetZLUT(0, 1, zplanes);
	if (cpucmp) qtrkcpu.SetZLUT(0, 1, zplanes);
	if (haveZLUT) {
		for (int x=0;x<zplanes;x++)  {
			vector2f center ( cfg.width/2, cfg.height/2 );
			float s = zmin + (zmax-zmin) * x/(float)(zplanes-1);
			GenerateTestImage(ImageData(image, cfg.width, cfg.height), center.x, center.y, s, 0.0f);
			FloatToJPEGFile("qtrkzlutimg.jpg", image, cfg.width,cfg.height);
			LocalizeType flags = (LocalizeType)(LocalizeBuildZLUT|LocalizeOnlyCOM);
			LocalizationJob jobInfo;
			jobInfo.frame = jobInfo.zlutPlane = x;
			jobInfo.locType = flags;
			jobInfo.zlutIndex = 0;
			qtrk.ScheduleLocalization((uchar*)image, cfg.width*sizeof(float),QTrkFloat, &jobInfo);
			if (cpucmp) qtrkcpu.ScheduleLocalization((uchar*)image, cfg.width*sizeof(float),QTrkFloat, &jobInfo);
		}
		qtrk.Flush();
		if (cpucmp) qtrkcpu.Flush();
		// wait to finish ZLUT
		while(true) {
			int rc = qtrk.GetResultCount();
			if (rc == zplanes) break;
			Sleep(100);
			dbgprintf(".");
		}
		if (cpucmp) {
			while(qtrkcpu.GetResultCount() != zplanes);
		}
	}
	float* zlut = qtrk.GetZLUT(0,0);
	if (cpucmp) { 
		float* zlutcpu = qtrkcpu.GetZLUT(0,0);

		WriteImageAsCSV("zlut-cpu.txt", zlutcpu, cfg.zlut_radialsteps, zplanes);
		WriteImageAsCSV("zlut-gpu.txt", zlut, cfg.zlut_radialsteps, zplanes);
	}
	qtrk.ClearResults();
	if (cpucmp) qtrkcpu.ClearResults();
	FloatToJPEGFile ("qtrkzlutcuda.jpg", zlut, cfg.zlut_radialsteps, zplanes);
	delete[] zlut;
	
	// Schedule images to localize on
	dbgprintf("Benchmarking...\n", total);
	GenerateTestImage(ImageData(image, cfg.width, cfg.height), cfg.width/2, cfg.height/2, (zmin+zmax)/2, 0);
	double tstart = GetPreciseTime();
	int rc = 0, displayrc=0;
	for (int n=0;n<total;n++) {
		LocalizeType flags = (LocalizeType)(LocalizeQI| (haveZLUT ? LocalizeZ : 0) );
		LocalizationJob jobInfo;
		jobInfo.frame = n;
		jobInfo.locType = flags;
		jobInfo.zlutIndex = 0;
		qtrk.ScheduleLocalization((uchar*)image, cfg.width*sizeof(float), QTrkFloat,&jobInfo);
		if (cpucmp) qtrkcpu.ScheduleLocalization((uchar*)image, cfg.width*sizeof(float), QTrkFloat, &jobInfo);
		if (n % 10 == 0) {
			rc = qtrk.GetResultCount();
			while (displayrc<rc) {
				if( displayrc%(total/10)==0) dbgprintf("Done: %d / %d\n", displayrc, total);
				displayrc++;
			}
		}
	}
	if (cpucmp) qtrkcpu.Flush();
	qtrk.Flush();
	do {
		rc = qtrk.GetResultCount();
		while (displayrc<rc) {
			if( displayrc%std::max(1,total/10)==0) dbgprintf("Done: %d / %d\n", displayrc, total);
			displayrc++;
		}
		Sleep(10);
	} while (rc != total);
	
	// Measure speed
	double tend = GetPreciseTime();

	if (cpucmp) {
		dbgprintf("waiting for cpu results..\n");
		while (total != qtrkcpu.GetResultCount())
			Sleep(10);
	}
	

	delete[] image;

	const int NumResults = 20;
	LocalizationResult results[NumResults], resultscpu[NumResults];
	int rcount = std::min(NumResults,total);
	for (int i=0;i<rcount;i++) {
		qtrk.PollFinished(&results[i], 1);
		if (cpucmp) qtrkcpu.PollFinished(&resultscpu[i], 1);
	}
	std::sort(results, results+rcount, [](LocalizationResult a, LocalizationResult b) -> bool { return a.job.frame > b.job.frame; });
	if(cpucmp) std::sort(resultscpu, resultscpu+rcount, [](LocalizationResult a, LocalizationResult b) -> bool { return a.job.frame > b.job.frame; });
	for (int i=0;i<rcount;i++) {
		LocalizationResult& r = results[i];
		dbgprintf("gpu [%d] x: %f, y: %f. z: %+g, COM: %f, %f\n", i,r.pos.x, r.pos.y, r.pos.z, r.firstGuess.x, r.firstGuess.y);

		if (cpucmp) {
			r = resultscpu[i];
			dbgprintf("cpu [%d] x: %f, y: %f. z: %+g, COM: %f, %f\n", i,r.pos.x, r.pos.y, r.pos.z, r.firstGuess.x, r.firstGuess.y);
		}
	}

	dbgprintf("Localization Speed: %d (img/s)\n", (int)( total/(tend-tstart) ));
}

void listDevices()
{
	hipDeviceProp_t prop;
	int dc;
	hipGetDeviceCount(&dc);
	for (int k=0;k<dc;k++) {
		hipGetDeviceProperties(&prop, k);
		dbgprintf("Device[%d] = %s\n", k, prop.name);
	}

}

__global__ void SimpleKernel(int N, float* a){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		for (int x=0;x<1000;x++)
			a[idx] = asin(a[idx]+x);
	}
}


void TestAsync()
{
	int N =100000;
	int nt = 32;

	pinned_array<float> a(N); 
//	hipHostMalloc(&a, sizeof(float)*N, 0);

	device_vec<float> A(N);

	hipStream_t s0, s1;
	hipEvent_t done;

	hipStreamCreate(&s0);
	hipEventCreate(&done,0);

	for (int x=0;x<N;x++)
		a[x] = cos(x*0.01f);

	for (int x=0;x<1;x++) {
		{ MeasureTime mt("a->A"); A.copyToDevice(a.data(), N, true); }
		{ MeasureTime mt("func(A)"); 
		SimpleKernel<<<dim3( (N+nt-1)/nt ), dim3(nt)>>>(N, A.data);
		}
		{ MeasureTime mt("A->a"); A.copyToHost(a.data(), true); }
	}
	hipEventRecord(done);

	{
	MeasureTime("sync..."); while (hipEventQuery(done) != hipSuccess); 
	}
	
	hipStreamDestroy(s0);
	hipEventDestroy(done);
}

__global__ void emptyKernel()
{}

float SpeedTest(const QTrkSettings& cfg, QueuedTracker* qtrk, int count, bool haveZLUT, LocalizeType locType)
{
	float *image = new float[cfg.width*cfg.height];
	srand(1);

	// Generate ZLUT
	int zplanes=100;
	float zmin=0.5,zmax=3;
	qtrk->SetZLUT(0, 1, zplanes);
	if (haveZLUT) {
		for (int x=0;x<zplanes;x++)  {
			vector2f center( cfg.width/2, cfg.height/2 );
			float s = zmin + (zmax-zmin) * x/(float)(zplanes-1);
			GenerateTestImage(ImageData(image, cfg.width, cfg.height), center.x, center.y, s, 0.0f);
			LocalizeType flags = (LocalizeType)(LocalizeBuildZLUT|LocalizeOnlyCOM);
			qtrk->ScheduleLocalization((uchar*)image, cfg.width*sizeof(float),QTrkFloat, flags , x, 0,0, 0, x);
		}
		qtrk->Flush();
		// wait to finish ZLUT
		while(true) {
			int rc = qtrk->GetResultCount();
			if (rc == zplanes) break;
			Sleep(100);
			dbgprintf(".");
		}
	}
	qtrk->ClearResults();
	
	// Schedule images to localize on
	dbgprintf("Benchmarking...\n", count);
	GenerateTestImage(ImageData(image, cfg.width, cfg.height), cfg.width/2, cfg.height/2, (zmin+zmax)/2, 0);
	double tstart = GetPreciseTime();
	int rc = 0, displayrc=0;
	double maxScheduleTime = 0.0f;
	double sumScheduleTime2 = 0.0f;
	double sumScheduleTime = 0.0f;
	for (int n=0;n<count;n++) {
		LocalizeType flags = (LocalizeType)(locType| (haveZLUT ? LocalizeZ : 0) );

		double t0 = GetPreciseTime();
		qtrk->ScheduleLocalization((uchar*)image, cfg.width*sizeof(float), QTrkFloat, flags, n, 0, 0, 0, 0);
		double dt = GetPreciseTime() - t0;
		maxScheduleTime = std::max(maxScheduleTime, dt);
		sumScheduleTime += dt;
		sumScheduleTime2 += dt*dt;

		if (n % 10 == 0) {
			rc = qtrk->GetResultCount();
			while (displayrc<rc) {
				if( displayrc%(count/10)==0) dbgprintf("Done: %d / %d\n", displayrc, count);
				displayrc++;
			}
		}
	}
	qtrk->Flush();
	do {
		rc = qtrk->GetResultCount();
		while (displayrc<rc) {
			if( displayrc%std::max(1,count/10)==0) dbgprintf("Done: %d / %d\n", displayrc, count);
			displayrc++;
		}
		Sleep(10);
	} while (rc != count);
	
	// Measure speed
	double tend = GetPreciseTime();
	delete[] image;

	float mean = sumScheduleTime / count;
	float stdev = sqrt(sumScheduleTime2 / count - mean * mean);
	dbgprintf("Scheduletime: Avg=%f, Max=%f, Stdev=%f\n", mean, maxScheduleTime, stdev);

	return count/(tend-tstart);
}

int NearestPowerOfTwo(int v)
{
	int r=1;
	while (r < v) 
		r *= 2;
	if ( fabsf(r-v) < fabsf(r/2-v) )
		return r;
	return r/2;
}

int SmallestPowerOfTwo(int minval)
{
	int r=1;
	while (r < minval)
		r *= 2;
	return r;
}



struct SpeedInfo {
	float cpu, gpu;
};

SpeedInfo SpeedCompareTest(int w)
{
	int cudaBatchSize = 1024;
	int count = 50000;

#ifdef _DEBUG
	count = 100;
	cudaBatchSize = 32;
#endif
	bool haveZLUT = false;
	LocalizeType locType = LocalizeQI;

	QTrkSettings cfg;
	cfg.width = cfg.height = w;
	cfg.qi_iterations = 4;
	cfg.qi_maxradius = cfg.width/2-8;
	//std::vector<int> devices(1); devices[0]=1;
	//SetCUDADevices(devices);
	cfg.cuda_device = QTrkCUDA_UseAll;
	cfg.qi_angsteps_per_quadrant = 32;
	cfg.qi_radialsteps = (int) (cfg.qi_maxradius-cfg.qi_minradius);
	cfg.numThreads = -1;
	cfg.com_bgcorrection = 0.0f;
	cfg.zlut_maxradius = 40;
	cfg.zlut_radialsteps = 64;
	cfg.zlut_angularsteps = 128;
	dbgprintf("Width: %d, QI radius: %f, radialsteps: %d\n", w, cfg.qi_maxradius, cfg.qi_radialsteps);

	QueuedCPUTracker *cputrk = new QueuedCPUTracker(&cfg);
	float cpuspeed = SpeedTest(cfg, cputrk, count, haveZLUT, locType);
	delete cputrk;

	QueuedCUDATracker *cudatrk = new QueuedCUDATracker(&cfg, cudaBatchSize);
	cudatrk->EnableTextureCache(true);
	float gpuspeed = SpeedTest(cfg, cudatrk, count, haveZLUT, locType);
	std::string report = cudatrk->GetProfileReport();
	delete cudatrk;

	auto profiling = QueuedCUDATracker::GetProfilingResults();
	for (auto i = profiling.begin(); i != profiling.end(); ++i) {
		auto r = i->second;
		dbgprintf("%s took %f ms on average\n", i->first, 1000*r.second/r.first);
	}

	dbgprintf("CPU tracking speed: %d img/s\n", (int)cpuspeed);
	dbgprintf("GPU tracking speed: %d img/s\n", (int)gpuspeed);
//	dbgout(report);

	SpeedInfo info;
	info.cpu = cpuspeed;
	info.gpu = gpuspeed;
	return info;
}

void ProfileSpeedVsROI()
{
	int N=24;
	float* values = new float[N*3];

	for (int i=0;i<N;i++) {
		int roi = 40+i*5;
		SpeedInfo info = SpeedCompareTest(roi);
		values[i*3+0] = info.cpu;
		values[i*3+1] = info.gpu;
	}

	const char *labels[] = { "CPU", "CUDA" };
	WriteImageAsCSV("speeds.txt", values, 2, N, labels);
	delete[] values;
}

std::vector<vector3f> LocalizeGeneratedImages(const QTrkSettings& cfg, QueuedTracker* qtrk, bool haveZLUT, LocalizeType locType, std::vector<vector3f> positions)
{
	float *image = new float[cfg.width*cfg.height];
	srand(1);

	// Generate ZLUT
	int zplanes=100;
	int count = positions.size();
	float zmin=0.5,zmax=3;
	qtrk->SetZLUT(0, 1, zplanes);
	if (haveZLUT) {
		for (int x=0;x<zplanes;x++)  {
			vector2f center( cfg.width/2, cfg.height/2 );
			float s = zmin + (zmax-zmin) * x/(float)(zplanes-1);
			GenerateTestImage(ImageData(image, cfg.width, cfg.height), center.x, center.y, s, 0.0f);
			LocalizeType flags = (LocalizeType)(LocalizeBuildZLUT|LocalizeQI);
			qtrk->ScheduleLocalization((uchar*)image, cfg.width*sizeof(float),QTrkFloat, flags , x, 0,0, 0, x);
		}
		qtrk->Flush();
		// wait to finish ZLUT
		while (qtrk->GetResultCount() != zplanes) {
			Sleep(100);
			dbgprintf(".");
		}
	}
	qtrk->ClearResults();
	int rc = 0, displayrc=0;
	for (int n=0;n<count;n++) {
		vector3f pos = positions[n];
		LocalizeType flags = (LocalizeType)(locType| (haveZLUT ? LocalizeZ : 0) );
		float s = zmin + (zmax-zmin) * pos.z/zplanes;
		GenerateTestImage(ImageData(image, cfg.width, cfg.height), cfg.width/2 + pos.x, cfg.height/2 + pos.y, s, 0);
		//if (n<5) FloatToJPEGFile(SPrintf("tracker-%d.jpg", n).c_str(), image, cfg.width,cfg.height);
		qtrk->ScheduleLocalization((uchar*)image, cfg.width*sizeof(float), QTrkFloat, flags, n, 0, 0, 0, 0);
	}
	qtrk->Flush();
	while (qtrk->GetResultCount() != count) Sleep(10);

	std::vector<LocalizationResult> results (count);
	qtrk->PollFinished( &results[0], count );
	std::sort (results.begin(), results.end(), 
		[](LocalizationResult& a, LocalizationResult& b) { return a.job.frame < b.job.frame; } );

	std::vector<vector3f> resultPos(count);
	for (int i=0;i<count;i++) {
		resultPos[i] = results[i].pos;
	}

	delete[] image;
	return resultPos;
}


void CompareAccuracy ()
{
	QTrkSettings cfg;
	cfg.width = cfg.height = 80;
	cfg.qi_iterations = 4;
	cfg.qi_maxradius = cfg.width/2-8;
	//std::vector<int> devices(1); devices[0]=1;
	//SetCUDADevices(devices);
	cfg.cuda_device = QTrkCUDA_UseAll;
	cfg.qi_angsteps_per_quadrant = 32;
	cfg.qi_radialsteps = NearestPowerOfTwo(cfg.qi_maxradius);
	cfg.numThreads = -1;
	cfg.com_bgcorrection = 0.0f;
	cfg.zlut_maxradius = cfg.qi_maxradius;
	cfg.zlut_radialsteps = 64;
	cfg.zlut_angularsteps = 128;

	int n = 5000;
#ifdef _DEBUG
	n = 2;
#endif
	bool haveZLUT = true;

	std::vector<vector3f> truePos (n);
	for (int i=0;i<n;i++) {
		vector3f p;
		p.x = 5 * ( rand_uniform<float>() - 0.5f );
		p.y = 5 * ( rand_uniform<float>() - 0.5f );
		p.z = 10 + 90 * rand_uniform<float>(); // 100 planes
		//p.z = 50;
		truePos [i] = p;
	}

	std::vector<QueuedTracker*> trackers;
	trackers.push_back (new QueuedCUDATracker(&cfg));
	((QueuedCUDATracker*) trackers.back())->EnableTextureCache(true);
	trackers.push_back (new QueuedCUDATracker(&cfg));
	((QueuedCUDATracker*) trackers.back())->EnableTextureCache(false);
	trackers.push_back(new QueuedCPUTracker(&cfg));

	auto results = new vector3f[ trackers.size() * n ];

	for (int i=0;i<trackers.size();i++) {
		double t0 = GetPreciseTime();
		auto r = LocalizeGeneratedImages(cfg, trackers[i], haveZLUT, LocalizeQI, truePos);
		for (int j=0;j<n;j++) 
			results[j * trackers.size() + i] = r[j];
		double t1 = GetPreciseTime();
		dbgprintf("tracker %d done. (%1.2f s) \n", i, t1-t0);
		//dbgout( trackers[i]->GetProfileReport() );
	}
	const char *labels[] = { "cudatcx","cudatcy","cudatcz", "cudax","cuday","cudaz", "cpux", "cpuy", "cpuz"};
	WriteImageAsCSV( "cmpresults.txt" , (float*)results, trackers.size()*3, n, labels );

	DeleteAllElems(trackers);
}


texture<float, hipTextureType2D, hipReadModeElementType> test_tex(0, hipFilterModePoint); // Un-normalized
texture<float, hipTextureType2D, hipReadModeElementType> test_tex_lin(0, hipFilterModeLinear); // Un-normalized


__global__ void TestSampling(int n , cudaImageListf img, float *rtex, float *rtex2, float *rmem, float2* pts)
{
	int idx = threadIdx.x+blockDim.x * blockIdx.x;

	if (idx < n) {
		float x = pts[idx].x;
		float y = pts[idx].y;
		int ii = 1;
		rtex[idx] = tex2D(test_tex_lin, x+0.5f, y+0.5f+img.h*ii);
		rtex2[idx] = img.interpolateFromTexture(test_tex, x, y, ii);
		rmem[idx] = img.interpolate(x,y,ii);
	}
}

void TestTextureFetch()
{
	int w=8,h=4;
	cudaImageListf img = cudaImageListf::alloc(w,h,2);
	float* himg = new float[w*h*2];

	int N=10;
	std::vector<vector2f> pts(N);
	for(int i=0;i<N;i++) {
		pts[i]=vector2f( rand_uniform<float>() * (w-1), rand_uniform<float>() * (h-1) );
	}
	device_vec<vector2f> dpts;
	dpts.copyToDevice(pts, false);

	srand(1);
	for (int i=0;i<w*h*2;i++)
		himg[i]=i;
	img.copyToDevice(himg,false);

	img.bind(test_tex);
	img.bind(test_tex_lin);
	device_vec<float> rtex(N),rmem(N),rtex2(N);
	int nt=32;
	TestSampling<<< dim3( (N+nt-1)/nt ), dim3(nt) >>> (N, img, rtex.data,rtex2.data,rmem.data, (float2*)dpts.data);
	img.unbind(test_tex_lin);
	img.unbind(test_tex);

	auto hmem = rmem.toVector();
	auto htex = rtex.toVector();
	auto htex2 = rtex2.toVector();
	for (int x=0;x<N;x++) {
		dbgprintf("[%.2f, %.2f]: %f (tex), %f(tex2),  %f (mem).  tex-mem: %f,  tex2-mem: %f\n",
		pts[x].x, pts[x].y, htex[x], htex2[x],	hmem[x],	htex[x]-hmem[x],htex2[x]-hmem[x]);
	}
}



int main(int argc, char *argv[])
{
	listDevices();
//	testLinearArray();

	//TestTextureFetch();

	CompareAccuracy();
	//QTrkTest();
	ProfileSpeedVsROI();
///	auto info = SpeedCompareTest(80);
	//dbgprintf("CPU: %f, GPU: %f, GPU(tc): %f\n", info.cpu, info.gpu, info.gputex); 
	return 0;
}
