#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "std_incl.h"
#include "utils.h"

#include <cassert>
#include <cstdlib>
#include <stdio.h>
#include <windows.h>
#include <cstdarg>
#include <valarray>

#include "cudafft/cudafft.h"
#include "random_distr.h"

#include <stdint.h>
#include "cudaImageList.h"
#include "QueuedCUDATracker.h"

#include <thrust/device_vector.h>

#define LSQFIT_FUNC __device__ __host__
#include "LsqQuadraticFit.h"

using namespace thrust;

double getPreciseTime()
{
	uint64_t freq, time;

	QueryPerformanceCounter((LARGE_INTEGER*)&time);
	QueryPerformanceFrequency((LARGE_INTEGER*)&freq);

	return (double)time / (double)freq;
}

std::string getPath(const char *file)
{
	std::string s = file;
	int pos = s.length()-1;
	while (pos>0 && s[pos]!='\\' && s[pos]!= '/' )
		pos--;
	
	return s.substr(0, pos);
}


texture<float, hipTextureType2D, hipReadModeElementType> xcor1D_images(0, hipFilterModeLinear);

inline __device__ float2 mul_conjugate(float2 a, float2 b)
{
	float2 r;
	r.x = a.x*b.x + a.y*b.y;
	r.y = a.y*b.x - a.x*b.y;
	return r;
}

template<typename T>
__device__ T max_(T a, T b) { return a>b ? a : b; }
template<typename T>
__device__ T min_(T a, T b) { return a<b ? a : b; }

template<typename T, int numPts>
__device__ T ComputeMaxInterp(T* data, int len)
{
	int iMax=0;
	T vMax=data[0];
	for (int k=1;k<len;k++) {
		if (data[k]>vMax) {
			vMax = data[k];
			iMax = k;
		}
	}
	T xs[numPts]; 
	int startPos = max_(iMax-numPts/2, 0);
	int endPos = min_(iMax+(numPts-numPts/2), len);
	int numpoints = endPos - startPos;


	if (numpoints<3) 
		return iMax;
	else {
		for(int i=startPos;i<endPos;i++)
			xs[i-startPos] = i-iMax;

		LsqSqQuadFit<T> qfit(numpoints, xs, &data[startPos]);
		//printf("iMax: %d. qfit: data[%d]=%f\n", iMax, startPos, data[startPos]);
		//for (int k=0;k<numpoints;k++) {
	//		printf("data[%d]=%f\n", startPos+k, data[startPos]);
		//}
		T interpMax = qfit.maxPos();

		if (fabs(qfit.a)<1e-9f)
			return (T)iMax;
		else
			return (T)iMax + interpMax;
	}
}

texture<float, hipTextureType2D, hipReadModeElementType> smpImgRef(0, hipFilterModeLinear);


__global__ void runCudaFFT(cudafft<float>::cpx_type *src, cudafft<float>::cpx_type *dst, cudafft<float>::KernelParams kparams)
{
	kparams.makeShared();
	cudafft<float>::transform(src,dst, kparams);
}



void TestKernelFFT()
{
	int N=256;
	cudafft<float> fft(N, false);

	std::vector< cudafft<float>::cpx_type > data(N), result(N), cpu_result(N);
	for (int x=0;x<N;x++)
		data[x].x = 10*cos(x*0.1f-5);

	fft.host_transform(&data[0], &cpu_result[0]);

	// now put data in video mem
	cudafft<float>::cpx_type *src,*d_result;
	int memSize = sizeof(cudafft<float>::cpx_type)*N;
	hipMalloc(&src, memSize);
	hipMemcpy(src, &data[0], memSize, hipMemcpyHostToDevice);
	hipMalloc(&d_result, memSize);

	int sharedMemSize = fft.kparams_size;
	for (int k=0;k<100;k++) {
		runCudaFFT<<<dim3(1),dim3(1),sharedMemSize>>>(src,d_result, fft.kparams);
	}

	hipMemcpy(&result[0], d_result, memSize, hipMemcpyDeviceToHost);

	for (int i=0;i<N;i++) {
		cudafft<float>::cpx_type v=cpu_result[i];
		cudafft<float>::cpx_type d=result[i];
		dbgprintf("[%d] CPU: %.1f+%.1fi. GPU: %.1f,%.1f\n", i, v.x, v.y, d.x,d.y);
	}

	hipFree(src);
	hipFree(d_result);
}

__global__ void test()
{
}


void ShowCUDAError() {
	hipError_t err = hipGetLastError();
	dbgprintf("Cuda error: %s\n", hipGetErrorString(err));
}

void testCOM()
{
	QTrkSettings cfg;
	cfg.numThreads = -1;
	QueuedCUDATracker trk(&cfg);

	cudaImageListf images = cudaImageListf::alloc(128,128,32);
	std::vector<float3> positions(images.count);

	for(int i=0;i<images.count;i++) {
		float xp = images.w/2+(rand_uniform<float>() - 0.5) * 5;
		float yp = images.h/2+(rand_uniform<float>() - 0.5) * 5;
		positions[i] = make_float3(xp, yp, 3);
		dbgprintf("Pos[%d]=( %f, %f )\n", i, xp, yp);
	}
//	device_vector<float3> d_pos(positions);
	float3* d_pos;
	hipMalloc(&d_pos, sizeof(float3)*images.count);
	hipMemcpy(d_pos, &positions[0], sizeof(float3)*images.count, hipMemcpyHostToDevice);

	test<<<dim3(),dim3()>>>();
	ShowCUDAError();
	//trk.GenerateImages(images, d_pos);
	images.free();
	hipFree(d_pos);
}

int main(int argc, char *argv[])
{
//	testLinearArray();

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);

	std::string path = getPath(argv[0]);

	testCOM();

	//TestKernelFFT();

	return 0;
}
