#include "hip/hip_runtime.h"
/*
Quadrant Interpolation on CUDA

Method:

-Batch images into host-side buffer
-Running batch:
	- Async copy host-side buffer to device
	- Bind image
	- Run COM kernel
	- QI loop: {
		- Run QI kernel: Sample from texture into quadrant profiles
		- Run CUFFT. Each iteration per axis does 2x forward FFT, and 1x backward FFT.
		- Run QI kernel: Compute positions
	}
	- Async copy results to host
	- Unbind image


Issues:
- Due to FPU operations on texture coordinates, there are small numerical differences between localizations of the same image at a different position in the batch
- 
*/

#include "std_incl.h"
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_vector_types.h"
#include <cstdint>
#include "utils.h"

#include "QueuedCUDATracker.h"
#include "simplefft.h"
#include "gpu_utils.h"

#define LSQFIT_FUNC __device__ __host__
#include "LsqQuadraticFit.h"

// Types used by QI algorithm
typedef float qivalue_t;
typedef sfft::complex<qivalue_t> qicomplex_t;


// According to this, textures bindings can be switched after the asynchronous kernel is launched
// https://devtalk.nvidia.com/default/topic/392245/texture-binding-and-stream/
texture<float, hipTextureType2D, hipReadModeElementType> qi_image_texture(0,  hipFilterModeLinear); // Un-normalized


__shared__ float2 cudaSharedMemory[];

// QueuedCUDATracker allows runtime choosing of GPU or CPU code. All GPU kernel calls are done through the following macro:
// Depending on 'useCPU' it either invokes a CUDA kernel named 'Funcname', or simply loops over the data on the CPU side calling 'Funcname' for each image
#define KERNEL_DISPATCH(Funcname, TParam) \
__global__ void Funcname##Kernel(cudaImageListf images, TParam param, int sharedMemPerThread) { \
	int idx = blockIdx.x * blockDim.x + threadIdx.x; \
	if (idx < images.count) { \
		Funcname(idx, images, &cudaSharedMemory [threadIdx.x * sharedMemPerThread], param); \
	} \
} \
void QueuedCUDATracker::CallKernel_##Funcname(cudaImageListf& images, TParam param, uint sharedMemPerThread)  { \
	Funcname##Kernel <<<blocks(images.count), threads(), sharedMemPerThread * numThreads >>> (images,param, sharedMemPerThread); \
}



QueuedTracker* CreateQueuedTracker(QTrkSettings* cfg)
{
	return new QueuedCUDATracker(cfg);
}

void CheckCUDAError()
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		const char* errstr = hipGetErrorString(err);
		dbgprintf("CUDA error: %s\n" ,errstr);
	}
}

QueuedCUDATracker::QueuedCUDATracker(QTrkSettings *cfg, int batchSize)
{
	this->cfg = *cfg;

	// Select the most powerful one
	if (cfg->cuda_device < 0) {
		int numDev;
		hipGetDeviceCount(&numDev);

		int bestScore;
		int bestDev;
		for (int a=0;a<numDev;a++) {
			int score;
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, a);
			score = prop.multiProcessorCount * prop.clockRate;
			if (a==0 || bestScore < score) {
				bestScore = score;
				bestDev = a;
			}
		}

		cfg->cuda_device = bestDev;
	}

	// We take numThreads to be the number of CUDA streams
	if (cfg->numThreads < 1) {
		cfg->numThreads = 4;
	}

	hipGetDeviceProperties(&deviceProp, cfg->cuda_device);
	numThreads = deviceProp.warpSize;
	
	if(batchSize<0) batchSize = 128;
	while (batchSize * cfg->height > deviceProp.maxTexture2D[1]) {
		batchSize/=2;
	}
	this->batchSize = batchSize;

	//int sharedSpacePerThread = (prop.sharedMemPerBlock-forward_fft->kparams_size*2) / numThreads;
//	dbgprintf("2X FFT instance requires %d bytes. Space per thread: %d\n", forward_fft->kparams_size*2, sharedSpacePerThread);
	dbgprintf("Device: %s\n", deviceProp.name);
	dbgprintf("Shared memory space:%d bytes. Per thread: %d\n", deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerBlock/numThreads);
	dbgprintf("# of CUDA processors:%d\n", deviceProp.multiProcessorCount);
	dbgprintf("Warp size: %d. Batch size: %d\n", deviceProp.warpSize, batchSize);

	qi_FFT_length = 1;
	while (qi_FFT_length < cfg->qi_radialsteps*2) qi_FFT_length *= 2;

	KernelParams &p = kernelParams;
	p.com_bgcorrection = cfg->com_bgcorrection;
	
	ZLUTParams& zp = p.zlut;
	zp.angularSteps = cfg->zlut_angularsteps;
	zp.maxRadius = cfg->zlut_maxradius;
	zp.minRadius = cfg->zlut_minradius;
	zp.planes = zlut_planes;

	QIParams& qi = p.qi;
	qi.angularSteps = cfg->qi_angularsteps;
	qi.iterations = cfg->qi_iterations;
	qi.maxRadius = cfg->qi_maxradius;
	qi.minRadius = cfg->qi_minradius;
	qi.radialSteps = cfg->qi_radialsteps;
	std::vector<float2> qiradialgrid(qi.angularSteps);
	for (int i=0;i<qi.angularSteps;i++)  {
		float ang = 0.5f*3.141593f*i/(float)qi.angularSteps;
		qiradialgrid[i]=make_float2(cos(ang), sin(ang));
	}
	d_qiradialgrid=qiradialgrid;
	qi.radialgrid=d_qiradialgrid.data;
	
	zlut = cudaImageListf::empty();
	kernelParams.zlut.img = zlut;

//	results.reserve(50000);
	
	streams.resize(cfg->numThreads);
	for (int i=0;i<streams.size();i++) {
		streams[i] = CreateStream();
	}
	currentStream=streams[0];
	int memUsePerStream = streams[0]->CalcMemoryUse();
	dbgprintf("Stream memory use: %d kb", memUsePerStream/1024);
}

QueuedCUDATracker::~QueuedCUDATracker()
{
	if (zlut.data)
		zlut.free();
	
	DeleteAllElems(streams);
}

__global__ void TestSampleImage(cudaImageListf images, int idx, float* dst)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x < images.w && y < images.h) {

		float p = tex2D( qi_image_texture, x, y);
		//float& p = images.pixel(x, y, idx);
		dst [y*images.w+x] = p;
	}
}

void TestCopyImage( cudaImageListf& images, int idx, const char* file)
{
	device_vec<float> ddst (images.w*images.h);
	int nt=32;
	hipDeviceSynchronize();
	TestSampleImage <<< dim3((images.w+nt-1)/nt, (images.h+nt-1)/nt), dim3(nt,nt) >>> (images, idx, ddst.data);

	std::vector<float> dst = ddst;
	uchar* norm = floatToNormalizedInt(&dst[0], images.w,images.h, (uchar)255);
	WriteJPEGFile(norm, images.w,images.h, file, 99);
	delete[] norm;
}


static __device__ float2 BgCorrectedCOM(int idx, cudaImageListf images, float correctionFactor)
{
	int imgsize = images.w*images.h;
	float sum=0, sum2=0;
	float momentX=0;
	float momentY=0;

	for (int y=0;y<images.h;y++)
		for (int x=0;x<images.w;x++) {
			float v = tex2D(qi_image_texture, x, y + idx*images.h);
			//float v = images.pixel(x,y,idx);
			sum += v;
			sum2 += v*v;
		}

	float invN = 1.0f/imgsize;
	float mean = sum * invN;
	float stdev = sqrtf(sum2 * invN - mean * mean);
	sum = 0.0f;

	for (int y=0;y<images.h;y++)
		for(int x=0;x<images.w;x++)
		{
			float v = tex2D(qi_image_texture, x, y + idx*images.h);
			//float v = images.pixel(x,y,idx);
			v = fabsf(v-mean)-correctionFactor*stdev;
			if(v<0.0f) v=0.0f;
			sum += v;
			momentX += (x+0.5f)*v;
			momentY += (y+0.5f)*v;
		}

	float2 com;
	com.x = momentX / (float)sum;
	com.y = momentY / (float)sum;
	return com;
}

__global__ void BgCorrectedCOM(int count, cudaImageListf images,float3* d_com, float bgCorrectionFactor) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < count) {
		float2 com = BgCorrectedCOM(idx, images, bgCorrectionFactor);
		d_com[idx] = make_float3(com.x,com.y,0.0f);
	}
}

static __device__ void RadialProfile(int idx, cudaImageListf& images, float *dst, const ZLUTParams zlut, float2 center, bool& error)
{
	int radialSteps = zlut.img.w;
	for (int i=0;i<zlut.img.w;i++)
		dst[i]=0.0f;

	float totalrmssum2 = 0.0f;
	float rstep = (zlut.maxRadius-zlut.minRadius) / radialSteps;
	for (int i=0;i<radialSteps; i++) {
		float sum = 0.0f;

		float r = zlut.minRadius+rstep*i;
		for (int a=0;a<zlut.angularSteps;a++) {
			float ang = 2*3.141593f*a/(float)zlut.angularSteps;
			float x = center.x + __cosf(ang) * r;
			float y = center.y + __sinf(ang) * r;
			sum += images.interpolate(x,y, idx);
		}

		dst[i] = sum/zlut.angularSteps-images.borderValue;
		totalrmssum2 += dst[i]*dst[i];
	}
	double invTotalrms = 1.0f/sqrt(totalrmssum2/radialSteps);
	for (int i=0;i<radialSteps;i++) {
		dst[i] *= invTotalrms;
	}
}

static __device__ qivalue_t QI_ComputeOffset(qicomplex_t* profile, qicomplex_t* tmpbuf1, const QIParams& params, int idx, sfft::complex<float>* s_twiddles) {
	int nr = params.radialSteps;

	qicomplex_t* reverse = tmpbuf1;

	for(int x=0;x<nr*2;x++)
		reverse[x] = profile[nr*2-1-x];

//	std::vector< sfft::complex<float> > tw = sfft::fill_twiddles<float> (nr*2);
	sfft::fft_forward(nr*2, profile, s_twiddles);
	sfft::fft_forward(nr*2, reverse, s_twiddles);

	// multiply with conjugate
	for(int x=0;x<nr*2;x++)
		profile[x] = profile[x] * reverse[x].conjugate();

	sfft::fft_inverse(nr*2, profile, s_twiddles);
	// fft_out2 now contains the autoconvolution
	// convert it to float
	qivalue_t* autoconv = (qivalue_t*)reverse;
	for(int x=0;x<nr*2;x++)  {
		autoconv[x] = profile[(x+nr)%(nr*2)].real();
	}

	float maxPos = ComputeMaxInterp<qivalue_t,7>(autoconv, nr*2);
	//free(reverse);
	return (maxPos - nr) / (3.14159265359f * 0.5f);
}


__global__ void ComputeQIKernel(cudaImageListf images, KernelParams params, float2* d_initial, float2* d_result)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < images.count) {
		bool error;
	//	d_result[idx] = ComputeQIPosition(idx, images, params, d_initial[idx], error);
	}
}


QueuedCUDATracker::Stream::Stream()
{ 
	hostImageBuf = 0; 
	images.data=0; 
	stream=0;
	state = StreamIdle;
	localizeFlags=0;
	jobCount = 0;
}

QueuedCUDATracker::Stream::~Stream() 
{
	hipStreamDestroy(stream);
	hipfftDestroy(fftPlan);

	if(images.data) images.free();
	hipEventDestroy(localizationDone);
}

bool QueuedCUDATracker::Stream::IsExecutionDone()
{
	return hipEventQuery(localizationDone) == hipSuccess;
}

int QueuedCUDATracker::Stream::CalcMemoryUse()
{
	return d_com.memsize() + d_jobs.memsize() + d_QIprofiles.memsize() + d_quadrants.memsize() + d_resultpos.memsize();
}

int QueuedCUDATracker::Stream::GetJobCount()
{
	mutex.lock();
	int jc = jobCount;
	mutex.unlock();
	return jc;
}

QueuedCUDATracker::Stream* QueuedCUDATracker::CreateStream()
{
	Stream* s = new Stream();

	hipStreamCreate(&s->stream);

	uint hostBufSize = sizeof(float)* cfg.width*cfg.height*batchSize;
	s->hostImageBuf.init(hostBufSize);
	s->images = cudaImageListf::alloc(cfg.width, cfg.height, batchSize);

	s->jobs.init(batchSize);
	s->results.init(batchSize);
	s->d_com.init(batchSize);
	s->d_resultpos.init(batchSize);
	s->results.init(batchSize);
	s->d_jobs.init(batchSize);
	s->d_quadrants.init(qi_FFT_length*batchSize*2);
	s->d_QIprofiles.init(batchSize*4*qi_FFT_length); // (2 axis) * (2 radialsteps) * (rev/forw) = 8 * nr = 4 * qi_FFT_length
	s->d_QIprofiles_reverse = s->d_QIprofiles.data + batchSize*2*qi_FFT_length;
	
	// 2* batchSize, since X & Y both need an FFT transform
	//hipfftResult_t r = hipfftPlanMany(&s->fftPlan, 1, &qi_FFT_length, 0, 1, qi_FFT_length, 0, 1, qi_FFT_length, HIPFFT_C2C, batchSize*4);
	hipfftResult_t r = hipfftPlan1d(&s->fftPlan, qi_FFT_length, HIPFFT_C2C, batchSize*2);

	if(r != HIPFFT_SUCCESS) {
		throw std::runtime_error( SPrintf("CUFFT plan creation failed. FFT len: %d. Batchsize: %d\n", qi_FFT_length, batchSize*4));
	}
	cufftSetCompatibilityMode(s->fftPlan, CUFFT_COMPATIBILITY_NATIVE);
	hipfftSetStream(s->fftPlan, s->stream);

	hipEventCreate(&s->localizationDone);
	return s;
}

 // get a stream that not currently executing, and still has room for images
QueuedCUDATracker::Stream* QueuedCUDATracker::GetReadyStream()
{
	if (currentStream && currentStream->state != Stream::StreamExecuting && 
		currentStream->GetJobCount() < batchSize) {
		return currentStream;
	}

	// Find another stream that is ready
	while (true) {
		FetchResults();
		for (int a=0;a<streams.size();a++) {
			Stream *s = streams[a];
			if (s->state != Stream::StreamExecuting) {
				currentStream = s;
				dbgprintf("Switching to stream %d\n", a);
				return s;
			}
		}
		Threads::Sleep(1);
	}
}



void QueuedCUDATracker::Start() 
{

}


void QueuedCUDATracker::ClearResults()
{
	FetchResults();
	results.clear();
}

// All streams on StreamIdle?
bool QueuedCUDATracker::IsIdle()
{
	return CheckAllStreams(Stream::StreamIdle);
}

bool QueuedCUDATracker::CheckAllStreams(Stream::State s)
{
	FetchResults();
	for (int a=0;a<streams.size();a++){
		if (streams[a]->state != s)
			return false;
	}
	return true;
}

bool QueuedCUDATracker::IsQueueFilled()
{
	return CheckAllStreams(Stream::StreamExecuting);
}

bool QueuedCUDATracker::ScheduleLocalization(uchar* data, int pitch, QTRK_PixelDataType pdt, LocalizeType locType, uint id, vector3f* initialPos, uint zlutIndex, uint zlutPlane)
{
	Stream* s = GetReadyStream();

	s->lock();

	int jobIndex =  s->jobCount++;
	CUDATrackerJob& job = s->jobs[jobIndex];
	if (initialPos)
		job.initialPos = *(float3*)initialPos;
	job.id = id;
	job.zlut = zlutIndex;
	job.locType = locType;
	job.zlutPlane = zlutPlane;
	s->localizeFlags |= locType; // which kernels to run

	// Copy the image to the batch image buffer (CPU side)
	float* hostbuf = &s->hostImageBuf[cfg.height*cfg.width*jobIndex];
	CopyImageToFloat(data, cfg.width, cfg.height, pitch, pdt, hostbuf);

//	tmp = floatToNormalizedInt( (float*)hostbuf, cfg.width,cfg.height,(uchar)255);
//	WriteJPEGFile(tmp, cfg.width,cfg.height, "writehostbuf2.jpg", 99);
//	delete[] tmp;

	// If batch is filled, copy the image to video memory asynchronously, and start the localization
	if (s->jobCount == batchSize)
		ExecuteBatch(s);

	s->unlock();

	return true;
}

/*
		q1: xprof[len-r-1], yprof[r]
		q2: xprof[len-r-1], yprof[len-r-1]
		q3: xprof[r], yprof[len-r-1]
		q0: xprof[r], yprof[r]
*/
static __device__ void QI_ComputeProfile2(cudaImageListf& images, int idx, float* dst, const QIParams& params, int quadrant, float2 center)
{
	const int qmat[] = {
		1, 1,
		-1, 1,
		-1, -1,
		1, -1 };
	int mx = qmat[2*quadrant+0];
	int my = qmat[2*quadrant+1];

	for (int i=0;i<params.radialSteps;i++)
		dst[i]=0.0f;
	
	double total = 0.0f;
	float rstep = (params.maxRadius - params.minRadius) / params.radialSteps;
	for (int i=0;i<params.radialSteps; i++) {
		double sum = 0.0f;
		float r = params.minRadius + rstep * i;

		for (int a=0;a<params.angularSteps;a++) {
			float ang = 0.5f*3.141593f*a/(float)params.angularSteps;
			float x = center.x + mx*params.radialgrid[a].x * r;
			float y = center.y + my*params.radialgrid[a].y * r;
			sum += images.interpolate(x, y, idx);
		}
		dst[i] = sum/params.angularSteps-images.borderValue;
		total += dst[i];
	}
}



static __device__ void ComputeQuadrantProfile(cudaImageListf& images, int idx, float* dst, const QIParams& params, int quadrant, float2 center)
{
	const int qmat[] = {
		1, 1,
		-1, 1,
		-1, -1,
		1, -1 };
	int mx = qmat[2*quadrant+0];
	int my = qmat[2*quadrant+1];

	for (int i=0;i<params.radialSteps;i++)
		dst[i]=0.0f;
	
	float sum2=0.0f;
	float total = 0.0f;
	float rstep = (params.maxRadius - params.minRadius) / params.radialSteps;
	for (int i=0;i<params.radialSteps; i++) {
		float sum = 0.0f;
		float r = params.minRadius + rstep * i;

		for (int a=0;a<params.angularSteps;a++) {
			//float ang = 0.5f*3.141593f*i/(float)params.angularSteps;
	//		float x = center.x + mx*cosf(ang) * r;
//			float y = center.y + my*sinf(ang) * r;
			float x = center.x + mx*params.radialgrid[a].x * r;
			float y = center.y + my*params.radialgrid[a].y * r;
			//float v = images.interpolate(x, y, idx);;
			//sum += v;
			sum += tex2D(qi_image_texture, x,y + idx*images.h);
//			printf("[%d] sum[%d,%d]:%f\n", idx, a, i, v);
	//		printf("%f; ", v);
		}
		//printf("\n");

		dst[i] = sum/params.angularSteps-images.borderValue;
		total += dst[i];
		sum2+=sum;
	}

//	printf("[%d] sum2:%f\n", idx,sum2);

}

__global__ void QI_ComputeProfile(int count, cudaImageListf images, float3* initial,float3* dstpos, float* quadrants, float2* profiles, float2* reverseProfiles, QIParams params)
{
//ComputeQuadrantProfile(cudaImageListf& images, int idx, float* dst, const QIParams& params, int quadrant, float2 center)
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < count) {
		int fftlen = params.radialSteps*2;
		float* img_qdr = &quadrants[ idx * params.radialSteps * 4 ];
		for (int q=0;q<4;q++)
			ComputeQuadrantProfile(images, idx, &img_qdr[q*params.radialSteps], params, q, make_float2(initial[idx].x, initial[idx].y));

		int nr = params.radialSteps;
		qicomplex_t* imgprof = (qicomplex_t*) &profiles[idx * fftlen*2];
		qicomplex_t* x0 = imgprof;
		qicomplex_t* x1 = imgprof + nr*1;
		qicomplex_t* y0 = imgprof + nr*2;
		qicomplex_t* y1 = imgprof + nr*3;

		qicomplex_t* revprof = (qicomplex_t*)&reverseProfiles[idx*fftlen*2];
		qicomplex_t* xrev = revprof;
		qicomplex_t* yrev = revprof + nr*2;

		float* q0 = &img_qdr[0];
		float* q1 = &img_qdr[nr];
		float* q2 = &img_qdr[nr*2];
		float* q3 = &img_qdr[nr*3];

		// Build Ix = qL(-r) || qR(r)
		// qL = q1 + q2   (concat0)
		// qR = q0 + q3   (concat1)
		for(int r=0;r<nr;r++) {
			x0[nr-r-1] = qicomplex_t(q1[r]+q2[r]);
			x1[r] = qicomplex_t(q0[r]+q3[r]);
		}
		// Build Iy = [ qB(-r)  qT(r) ]
		// qT = q0 + q1
		// qB = q2 + q3
		for(int r=0;r<nr;r++) {
			y1[r] = qicomplex_t(q0[r]+q1[r]);
			y0[nr-r-1] = qicomplex_t(q2[r]+q3[r]);
		}


		for(int r=0;r<nr*2;r++)
			xrev[r] = x0[nr*2-r-1];
		for(int r=0;r<nr*2;r++)
			yrev[r] = y0[nr*2-r-1];
	}
}


__global__ void QI_MultiplyWithConjugate(int n, hipfftComplex* a, hipfftComplex* b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		hipfftComplex A = a[idx];
		hipfftComplex B = b[idx];
	
		a[idx] = make_float2(A.x*B.x + A.y*B.y, A.y*B.x -A.x*B.y); // multiplying with conjugate
	}
}

__shared__ float compute_offset_buffer[];

__device__ float QI_ComputeAxisOffset(hipfftComplex* autoconv, int fftlen)
{
	float* shifted = &compute_offset_buffer [threadIdx.x * fftlen];
	int nr = fftlen/2;
	for(int x=0;x<fftlen;x++)  {
		shifted[x] = autoconv[(x+nr)%(nr*2)].x;
	}

	float maxPos = ComputeMaxInterp<float,7>(shifted, fftlen);
	float offset = (maxPos - nr) / (3.14159265359f * 0.5f);
	return offset;
}

__global__ void QI_OffsetPositions(int njobs, float3* current, float3* dst, hipfftComplex* autoconv, int fftLength, float2* offsets)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < njobs) {
		// X
		hipfftComplex* autoconvX = &autoconv[idx * fftLength * 2];
		float xoffset = QI_ComputeAxisOffset(autoconvX, fftLength);

		hipfftComplex* autoconvY = autoconvX + fftLength;
		float yoffset = QI_ComputeAxisOffset(autoconvY, fftLength);

		dst[idx].x = current[idx].x + xoffset;
		dst[idx].y = current[idx].y + yoffset;

		if (offsets) 
			offsets[idx] = make_float2( xoffset, yoffset);
	}
}

static unsigned long hash(unsigned char *str, int n)
{
    unsigned long hash = 5381;
    
    for (int i=0;i<n;i++) {
		int c = str[i];
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
	}

    return hash;
}

template<typename T>
void checksum(T* data, int elemsize, int numelem, const char *name)
{
#ifdef _DEBUG
	uchar* cp = (uchar*)ALLOCA(elemsize*numelem*sizeof(T));
	hipDeviceSynchronize();
	hipMemcpy(cp, data, sizeof(T)*elemsize*numelem, hipMemcpyDeviceToHost);

	dbgprintf("%s:\n", name);
	for (int i=0;i<numelem;i++) {
		uchar *elem = cp+elemsize*sizeof(T)*i;
		dbgprintf("[%d]: %d\n", i, hash(elem, elemsize));
		for (int j=0;j<elemsize/4;j++) {
//			float* d = (float*)elem;

		}
	}
#endif
}

void QueuedCUDATracker::QI_Iterate(device_vec<float3>* initial, device_vec<float3>* newpos, Stream *s)
{
/*	int njobs = s->jobs.size();
	int nElem = njobs * qi_FFT_length * 2; // 2 profiles of qi_FFT_length size for each job
	QI_ComputeProfile <<< blocks(nElem), threads(), 0, s->stream >>> (qi_FFT_length, s->*/

//	hipDeviceSynchronize();
//	std::vector<float3> initial_h=*initial;

//	TestCopyImage(s->images, 0, "testimg.jpg");

	int njobs = s->jobCount;
	QI_ComputeProfile <<< blocks(njobs), threads(), 0, s->stream >>> (njobs, s->images, initial->data, newpos->data, 
		s->d_quadrants.data, s->d_QIprofiles.data, s->d_QIprofiles_reverse, kernelParams.qi);

	checksum(s->d_quadrants.data, qi_FFT_length * 2, njobs, "quadrant");
	checksum(s->d_QIprofiles.data, qi_FFT_length * 2, njobs, "prof");
	checksum(s->d_QIprofiles_reverse, qi_FFT_length * 2, njobs, "revprof");

#ifdef QI_DBG_EXPORT
	hipDeviceSynchronize();
	std::vector<float> hquadrants = s->d_quadrants;
	WriteImageAsCSV("quadrants.txt", (float*)&hquadrants[0], qi_FFT_length*2, njobs);
#endif

	hipfftComplex* prof = (hipfftComplex*)s->d_QIprofiles.data;
	hipfftComplex* revprof = s->d_QIprofiles_reverse;
#ifdef QI_DBG_EXPORT
	hipDeviceSynchronize();
	std::vector<float2> hprof = s->d_QIprofiles;
	WriteComplexImageAsCSV("profiles.txt", (std::complex<float>*)&hprof[0], qi_FFT_length*2, njobs*2);
#endif

	hipfftExecC2C(s->fftPlan, prof, prof, HIPFFT_FORWARD);
	hipfftExecC2C(s->fftPlan, revprof, revprof, HIPFFT_FORWARD);
#ifdef QI_DBG_EXPORT
	hipDeviceSynchronize();
	std::vector<float2> hprof_fft = s->d_QIprofiles;
	WriteComplexImageAsCSV("fftprofiles.txt", (std::complex<float>*)&hprof_fft[0], qi_FFT_length*2, njobs);
#endif

	int nval = qi_FFT_length * 2 * batchSize, nthread=256;
	QI_MultiplyWithConjugate<<< dim3( (nval + nthread - 1)/nthread ), dim3(nthread), 0, s->stream >>>(nval, prof, revprof);
	hipfftExecC2C(s->fftPlan, prof, prof, HIPFFT_BACKWARD);

	QI_OffsetPositions<<<blocks(njobs), threads(), sizeof(float)*qi_FFT_length*numThreads , s->stream>>>(njobs, initial->data, newpos->data, prof, qi_FFT_length, 0);
#ifdef QI_DBG_EXPORT
	device_vec<float2> offsets(njobs);
	QI_OffsetPositions<<<blocks(njobs), threads(), sizeof(float)*qi_FFT_length*numThreads, s->stream>>>(njobs, initial->data, newpos->data, prof, qi_FFT_length, offsets.data);
	hipDeviceSynchronize();
	std::vector<float2> h_offsets = offsets;
	for (int i=0;i<njobs;i++) {
		dbgprintf("Offset[%d]: x: %f, y: %f\n", i, h_offsets[i].x, h_offsets[i].y);
	}
	std::vector<float2> qiprof = s->d_QIprofiles;
	WriteComplexImageAsCSV("autoconv.txt", (std::complex<float>*)&qiprof[0], qi_FFT_length*2, njobs);
#endif
}


__global__ void BuildZLUTKernel(int njobs, cudaImageListf images, ZLUTParams params, float3* positions, CUDATrackerJob* jobs)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < njobs) {
		CUDATrackerJob& j = jobs[idx];
		if (j.locType & LocalizeBuildZLUT) {
			bool err;
			RadialProfile(idx, images, params.GetZLUT( j.zlut, j.zlutPlane ), params, make_float2(positions[idx].x, positions[idx].y), err);
		}
	}
}


void QueuedCUDATracker::ExecuteBatch(Stream *s)
{
	if (s->jobCount==0)
		return;
#ifdef _DEBUG
	dbgprintf("Sending %d images to GPU...\n", s->jobs.size());
#endif

/*	- Async copy host-side buffer to device
	- Bind image
	- Run COM kernel
	- QI loop: {
		- Run QI kernel: Sample from texture into quadrant profiles
		- Run CUFFT
		- Run QI kernel: Compute positions
	}
	- Async copy results to host
	- Unbind image
	*/

	hipMemcpy2DAsync( s->images.data, s->images.pitch, s->hostImageBuf.data(), sizeof(float)*s->images.w, s->images.w*sizeof(float), s->images.h * s->jobCount, hipMemcpyHostToDevice, s->stream);
	s->d_jobs.copyToDevice(s->jobs.data(), s->jobCount, true, s->stream);
	s->images.bind(qi_image_texture);
	BgCorrectedCOM <<< blocks(s->jobCount), threads(), 0, s->stream >>> (s->jobCount, s->images, s->d_com.data, cfg.com_bgcorrection);

	checksum(s->d_com.data, 2, s->jobCount, "com");

	device_vec<float3> *curpos = &s->d_com;
	for (int a=0;a<cfg.qi_iterations;a++) {
		QI_Iterate(curpos, &s->d_resultpos, s);
		curpos = &s->d_resultpos;
	}

	if (s->localizeFlags & LocalizeBuildZLUT) {
		BuildZLUTKernel <<< blocks(s->jobs.size()), threads(), 0, s->stream >>> (s->jobCount, s->images, kernelParams.zlut, curpos->data, s->d_jobs.data);
	//	TestCopyImage( s->images, 0, "qtrktestimg0.jpg");
	}
	
	curpos->copyToHost(s->results.data(), true, s->stream);
	
	s->images.unbind(qi_image_texture);
	//CheckCUDAError();
	
	// Make sure we can query the all done signal
	hipEventRecord(s->localizationDone);

	s->state = Stream::StreamExecuting;
}

void QueuedCUDATracker::Flush()
{
	if (currentStream) {
		currentStream->lock();
		ExecuteBatch(currentStream);
		currentStream->unlock();
	}
}

int QueuedCUDATracker::FetchResults()
{
	// Labview can call from multiple threads
	for (int a=0;a<streams.size();a++) {
		Stream* s = streams[a];
		if (s->state == Stream::StreamExecuting && s->IsExecutionDone()) {
			s->lock();
			CopyStreamResults(s);
			s->state = Stream::StreamIdle;
			s->unlock();
		}
	}
	return results.size();
}

void QueuedCUDATracker::CopyStreamResults(Stream *s)
{
	for (int a=0;a<s->jobCount;a++) {
		CUDATrackerJob& j = s->jobs[a];

		LocalizationResult r;
		r.error = j.error;
		r.id = j.id;
		r.firstGuess = vector2f();
		r.locType = j.locType;
		r.zlutIndex = j.zlut;
		r.pos.x = s->results[a].x;
		r.pos.y = s->results[a].y;
		r.z = s->results[a].z;

		results.push_back(r);
	}
	s->jobCount=0;
	s->localizeFlags = 0; // reset this for the next batch
}

int QueuedCUDATracker::PollFinished(LocalizationResult* dstResults, int maxResults)
{
	FetchResults();

	int numResults = 0;
	while (numResults < maxResults && !results.empty()) {
		dstResults[numResults++] = results.front();
		results.pop_front();
	}
	return numResults;
}

// data can be zero to allocate ZLUT data
void QueuedCUDATracker::SetZLUT(float* data,  int numLUTs, int planes, int res, float* zcmp) 
{
	zlut_planes = planes;
	zlut_count = numLUTs;
	zlut_res = res;

	if (zcmp) {
		zcompareWindow.copyToDevice(zcmp, res, false);
		kernelParams.zlut.zcmpwindow = zcompareWindow.data;
	}

	zlut = cudaImageListf::alloc(res, planes, numLUTs);
	if (data) zlut.copyToDevice(data, false);
	kernelParams.zlut.img = zlut;
}

// delete[] memory afterwards
float* QueuedCUDATracker::GetZLUT(int *count, int* planes, int *res)
{
	float* data = new float[zlut_planes * zlut_res * zlut_count];
	if (zlut.data)
		zlut.copyToHost(data, false);
	else
		std::fill(data, data+(zlut_res*zlut_planes*zlut_count), 0.0f);

	if (planes) *planes = zlut_planes;
	if (res) *res = zlut_res;
	if (count) *count = zlut_count;

	return data;
}


int QueuedCUDATracker::GetResultCount()
{
	return FetchResults();
}



// TODO: Let GPU copy frames from frames to GPU 
void QueuedCUDATracker::ScheduleFrame(uchar *imgptr, int pitch, int width, int height, ROIPosition *positions, int numROI, QTRK_PixelDataType pdt, 
									LocalizeType locType, uint frame, uint zlutPlane, bool async)
{
	uchar* img = (uchar*)imgptr;
	int bpp = sizeof(float);
	if (pdt == QTrkU8) bpp = 1;
	else if (pdt == QTrkU16) bpp = 2;
	for (int i=0;i<numROI;i++){
		uchar *roiptr = &img[pitch * positions[i].y + positions[i].x * bpp];
		ScheduleLocalization(roiptr, pitch, pdt, locType, frame, 0, i, zlutPlane);
	}
}

void QueuedCUDATracker::WaitForScheduleFrame(uchar* imgptr) {
}
