#include "hip/hip_runtime.h"
/*
CUDA implementations of a variety of tracking algorithms: COM, Quadrant Interpolation, 2D Gaussian with Max-Likelihood estimation.
Copyright 2012-2013, Jelmer Cnossen

It will automatically use all available CUDA devices if using the QTrkCUDA_UseAll value for QTrkSettings::cuda_device

Method:

-Load images into host-side image buffer
-Scheduling thread executes any batch that is filled

- Mutexes:
	* JobQueueMutex: controlling access to state and jobs. 
		Used by ScheduleLocalization, scheduler thread, and GetQueueLen
	* ResultMutex: controlling access to the results list, 
		locked by the scheduler whenever results come available, and by calling threads when they run GetResults/Count

-Running batch:
	- Async copy host-side buffer to device
	- Bind image
	- Run COM kernel
	- QI loop: {
		- Run QI kernel: Sample from texture into quadrant profiles
		- Run CUFFT. Each iteration per axis does 2x forward FFT, and 1x backward FFT.
		- Run QI kernel: Compute positions
	}
	- Compute ZLUT profiles
	- Depending on localize flags:
		- copy ZLUT profiles (for ComputeBuildZLUT flag)
		- generate compare profile kernel + compute Z kernel (for ComputeZ flag)
	- Unbind image
	- Async copy results to host

Issues:
- Due to FPU operations on texture coordinates, there are small numerical differences between localizations of the same image at a different position in the batch
*/
#include "std_incl.h"
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_vector_types.h"
#include <cstdint>
#include "utils.h"

#include "QueuedCUDATracker.h"
#include "gpu_utils.h"
#include "ImageSampler.h"

#define LSQFIT_FUNC __device__ __host__
#include "LsqQuadraticFit.h"

#include "Kernels.h"
#include "DebugResultCompare.h"


#ifdef QI_DEBUG
void DbgCopyResult(device_vec<float2>& src, std::vector< std::complex<float> >& dst) {
	hipDeviceSynchronize();
	std::vector<float2> x(src.size);
	src.copyToHost(x,false,0);
	dst.resize(src.size);
	for(int i=0;i<x.size();i++)
		dst[i]=std::complex<float>(x[i].x,x[i].y);
}
void DbgCopyResult(device_vec<float>& src, std::vector< float >& dst) {
	hipDeviceSynchronize();
	src.copyToHost(dst,false,0);
}
#else
void DbgCopyResult(device_vec<float2> src, std::vector< std::complex<float> >& dst) {} 
void DbgCopyResult(device_vec<float> src, std::vector< float>& dst) {}
#endif

// Do CPU-side profiling of kernel launches?
#define TRK_PROFILE

#ifdef TRK_PROFILE
	class ScopedCPUProfiler
	{
		double* time;
		double start;
	public:
		typedef std::pair<int, double> Item;
		static std::map<const char*, Item> results;

		ScopedCPUProfiler(double *time) :  time(time) {
			start = GetPreciseTime();
		}
		~ScopedCPUProfiler() {
			double end = GetPreciseTime();
			*time += start-end;
		}
	};
#else
	class ScopedCPUProfiler {
	public:
		ScopedCPUProfiler(double* time) {}
	};
#endif

static std::vector<int> cudaDeviceList; 

void SetCUDADevices(int* dev, int numdev) {
	cudaDeviceList.assign(dev,dev+numdev);
}



QueuedTracker* CreateQueuedTracker(const QTrkComputedConfig& cc)
{
	return new QueuedCUDATracker(cc);
}

void CheckCUDAError()
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		const char* errstr = hipGetErrorString(err);
		dbgprintf("CUDA error: %s\n" ,errstr);
	}
}

static int GetBestCUDADevice()
{
	int bestScore;
	int bestDev;
	int numDev;
	hipGetDeviceCount(&numDev);
	for (int a=0;a<numDev;a++) {
		int score;
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, a);
		score = prop.multiProcessorCount * prop.clockRate;
		if (a==0 || bestScore < score) {
			bestScore = score;
			bestDev = a;
		}
	}
	return bestDev;
}

void QueuedCUDATracker::InitializeDeviceList()
{
	int numDevices;
	hipGetDeviceCount(&numDevices);

	// Select the most powerful one
	if (cfg.cuda_device == QTrkCUDA_UseBest) {
		cfg.cuda_device = GetBestCUDADevice();
		devices.push_back(new Device(cfg.cuda_device));
	} else if(cfg.cuda_device == QTrkCUDA_UseAll) {
		// Use all devices
		for (int i=0;i<numDevices;i++)
			devices.push_back(new Device(i));
	} else if (cfg.cuda_device == QTrkCUDA_UseList) {
		for (uint i=0;i<cudaDeviceList.size();i++)
			devices.push_back(new Device(cudaDeviceList[i]));
	} else {
		devices.push_back (new Device(cfg.cuda_device));
	}
	deviceReport = "Using devices: ";
	for (uint i=0;i<devices.size();i++) {
		hipDeviceProp_t p; 
		hipGetDeviceProperties(&p, devices[i]->index);
		deviceReport += SPrintf("%s%s", p.name, i<devices.size()-1?", ":"\n");
	}
}


QueuedCUDATracker::QueuedCUDATracker(const QTrkComputedConfig& cc, int batchSize) 
	: resultMutex("result"), jobQueueMutex("jobqueue")
{
	cfg = cc;

	InitializeDeviceList();

	// We take numThreads to be the number of CUDA streams
	if (cfg.numThreads < 1) {
		cfg.numThreads = devices.size()*4;
	}
	int numStreams = cfg.numThreads;

	hipGetDeviceProperties(&deviceProp, devices[0]->index);
	numThreads = deviceProp.warpSize;
	
	if(batchSize<0) batchSize = 256;
	while (batchSize * cfg.height > deviceProp.maxTexture2D[1]) {
		batchSize/=2;
	}
	this->batchSize = batchSize;

	dbgprintf("# of CUDA processors:%d. Using %d streams\n", deviceProp.multiProcessorCount, numStreams);
	dbgprintf("Warp size: %d. Max threads: %d, Batch size: %d\n", deviceProp.warpSize, deviceProp.maxThreadsPerBlock, batchSize);

	KernelParams &p = kernelParams;
	p.com_bgcorrection = cfg.com_bgcorrection;
	
	ZLUTParams& zp = p.zlut;
	zp.angularSteps = cfg.zlut_angularsteps;
	zp.maxRadius = cfg.zlut_maxradius;
	zp.minRadius = cfg.zlut_minradius;
	zp.planes = 0;
	zp.zcmpwindow = 0;
	
	std::vector<float2> zlut_radialgrid(cfg.zlut_angularsteps);
	for (int i=0;i<cfg.zlut_angularsteps;i++) {
		float ang = 2*3.141593f*i/(float)cfg.zlut_angularsteps;
		zlut_radialgrid[i]=make_float2(cos(ang),sin(ang));
	}

	for (uint i=0;i<devices.size();i++) {
		Device* d = devices[i];
		hipSetDevice(d->index);
		qi.InitDevice(&d->qi_instance, cfg);
		d->zlut_trigtable = zlut_radialgrid;
	}
	kernelParams.zlut.img = cudaImageListf::emptyList();
	
	streams.reserve(numStreams);
	try {
		for (int i=0;i<numStreams;i++)
			streams.push_back( CreateStream( devices[i%devices.size()], i ) );
	}
	catch(...) {
		DeleteAllElems(streams);
		throw;
	}

	streams[0]->OutputMemoryUse();

	batchesDone = 0;
	useTextureCache = true;
	resultCount = 0;

	quitScheduler = false;
	schedulingThread = Threads::Create(SchedulingThreadEntryPoint, this);

	gc_offsetFactor = gc_gainFactor = 1.0f;

	imap_w = imap_h = imap_planes = 0;
}

QueuedCUDATracker::~QueuedCUDATracker()
{
	quitScheduler = true;
	Threads::WaitAndClose(schedulingThread);

	DeleteAllElems(streams);
	DeleteAllElems(devices);
}

QueuedCUDATracker::Device::~Device()
{
	hipSetDevice(index);
	radial_zlut.free();
	calib_gain.free();
	calib_offset.free();
	image_zlut.free();
}

void QueuedCUDATracker::SchedulingThreadEntryPoint(void *param)
{
	((QueuedCUDATracker*)param)->SchedulingThreadMain();
}

void QueuedCUDATracker::SchedulingThreadMain()
{
	std::vector<Stream*> activeStreams;

	while (!quitScheduler) {
		jobQueueMutex.lock();
		Stream* s = 0;
		for (int i=0;i<streams.size();i++) 
			if (streams[i]->state == Stream::StreamPendingExec) {
				s=streams[i];
				s->state = Stream::StreamExecuting;
			//	dbgprintf("Executing stream %p [%d]. %d jobs\n", s, i, s->JobCount());
				break;
			}
		jobQueueMutex.unlock();

		if (s) {
			s->imageBufMutex.lock();

			// Launch filled batches, or if flushing launch every batch with nonzero jobs
			if (useTextureCache)
				ExecuteBatch<ImageSampler_Tex> (s);
			else
				ExecuteBatch<ImageSampler_MemCopy> (s);
			s->imageBufMutex.unlock();
			activeStreams.push_back(s);
		}

		// Fetch results
		for (int a=0;a<activeStreams.size();a++) {
			Stream* s = activeStreams[a];
			if (s->IsExecutionDone()) {
		//		dbgprintf("Stream %p done.\n", s);
				CopyStreamResults(s);
				s->localizeFlags = 0; // reset this for the next batch
				jobQueueMutex.lock();
				s->jobs.clear();
				s->state = Stream::StreamIdle;
				jobQueueMutex.unlock();
				activeStreams.erase(std::find(activeStreams.begin(),activeStreams.end(),s));
				break;
			}
		}

		Threads::Sleep(1);
	}
}


QueuedCUDATracker::Stream::Stream(int streamIndex)
	: imageBufMutex(SPrintf("imagebuf%d", streamIndex).c_str())
{ 
	device = 0;
	hostImageBuf = 0; 
	images.data=0; 
	stream=0;
	state=StreamIdle;
	localizeFlags=0;
}

QueuedCUDATracker::Stream::~Stream() 
{
	hipSetDevice(device->index);

	if(images.data) images.free();
	hipEventDestroy(localizationDone);
	hipEventDestroy(qiDone);
	hipEventDestroy(comDone);
	hipEventDestroy(imageCopyDone);
	hipEventDestroy(zcomputeDone);
	hipEventDestroy(imapDone);
	hipEventDestroy(batchStart);

	if (stream)
		hipStreamDestroy(stream); // stream can be zero if in debugStream mode.
}


bool QueuedCUDATracker::Stream::IsExecutionDone()
{
	hipSetDevice(device->index);
	return hipEventQuery(localizationDone) == hipSuccess;
}


void QueuedCUDATracker::Stream::OutputMemoryUse()
{
	int deviceMem = d_com.memsize() + d_locParams.memsize() + qi_instance.memsize() + d_radialprofiles.memsize() +
		d_resultpos.memsize() + d_zlutcmpscores.memsize() + images.totalNumBytes();

	int hostMem = hostImageBuf.memsize() + com.memsize() + locParams.memsize() + results.memsize();

	dbgprintf("Stream memory use: %d kb pinned on host, %d kb device memory (%d for images). \n", hostMem / 1024, deviceMem/1024, images.totalNumBytes()/1024);
}


QueuedCUDATracker::Stream* QueuedCUDATracker::CreateStream(Device* device, int streamIndex)
{
	Stream* s = new Stream(streamIndex);

	try {
		s->device = device;
		hipSetDevice(device->index);
		hipStreamCreate(&s->stream);

		s->images = cudaImageListf::alloc(cfg.width, cfg.height, batchSize);
		s->images.allocateHostImageBuffer(s->hostImageBuf);

		s->jobs.reserve(batchSize);
		s->results.init(batchSize);
		s->com.init(batchSize);
		s->d_com.init(batchSize);
		s->d_resultpos.init(batchSize);
		s->results.init(batchSize);
		s->locParams.init(batchSize);
		s->d_imgmeans.init(batchSize);
		s->d_locParams.init(batchSize);
		s->d_radialprofiles.init(cfg.zlut_radialsteps*batchSize);

		qi.InitStream(&s->qi_instance, cfg, s->stream, batchSize);

		hipEventCreate(&s->localizationDone);
		hipEventCreate(&s->comDone);
		hipEventCreate(&s->imageCopyDone);
		hipEventCreate(&s->zcomputeDone);
		hipEventCreate(&s->qiDone);
		hipEventCreate(&s->imapDone);
		hipEventCreate(&s->batchStart);
	} catch (...) {
		delete s;
		throw;
	}
	return s;
}


 // get a stream that is not currently executing, and still has room for images
QueuedCUDATracker::Stream* QueuedCUDATracker::GetReadyStream()
{
	while (true) {
		jobQueueMutex.lock();
		
		Stream *best = 0;
		for (int i=0;i<streams.size();i++) 
		{
			Stream*s = streams[i];

			if (s->state == Stream::StreamIdle) {
				if (!best || (s->JobCount() > best->JobCount()))
					best = s;
			}
		}

		jobQueueMutex.unlock();

		if (best) 
			return best;

		Threads::Sleep(1);
	}
}


bool QueuedCUDATracker::IsIdle()
{
	int ql = GetQueueLength(0);
	return ql == 0;
}

int QueuedCUDATracker::GetQueueLength(int *maxQueueLen)
{
	jobQueueMutex.lock();
	int qlen = 0;
	for (uint a=0;a<streams.size();a++){
		qlen += streams[a]->JobCount();
	}
	jobQueueMutex.unlock();

	if (maxQueueLen) {
		*maxQueueLen = streams.size()*batchSize;
	}

	return qlen;
}


void QueuedCUDATracker::ScheduleLocalization(uchar* data, int pitch, QTRK_PixelDataType pdt, const LocalizationJob* jobInfo )
{
	Stream* s = GetReadyStream();

	jobQueueMutex.lock();
	int jobIndex = s->jobs.size();
	LocalizationJob job = *jobInfo;
	job.locType = jobInfo->LocType();
	if (s->device->radial_zlut.isEmpty())  // dont do ZLUT commands when no ZLUT has been set
		job.locType &= ~(LT_LocalizeZ | LT_BuildRadialZLUT);
	s->jobs.push_back(job);
	s->localizeFlags |= job.locType; // which kernels to run
	s->locParams[jobIndex].locType = job.LocType();
	s->locParams[jobIndex].zlutIndex = jobInfo->zlutIndex;
	s->locParams[jobIndex].zlutPlane = jobInfo->zlutPlane;

	if (s->jobs.size() == batchSize)
		s->state = Stream::StreamPendingExec;
	jobQueueMutex.unlock();

	s->imageBufMutex.lock();
	// Copy the image to the batch image buffer (CPU side)
	float* hostbuf = &s->hostImageBuf[cfg.height*cfg.width*jobIndex];
	CopyImageToFloat(data, cfg.width, cfg.height, pitch, pdt, hostbuf);
	s->imageBufMutex.unlock();

	//dbgprintf("Job: %d\n", jobIndex);
}


void QueuedCUDATracker::Flush()
{
	jobQueueMutex.lock();
	for (int i=0;i<streams.size();i++) {
		if(streams[i]->JobCount()>0 && streams[i]->state != Stream::StreamExecuting)
			streams[i]->state = Stream::StreamPendingExec;
	}
	jobQueueMutex.unlock();
}


#ifdef QI_DBG_EXPORT
static unsigned long hash(unsigned char *str, int n)
{
    unsigned long hash = 5381;
    
    for (int i=0;i<n;i++) {
		int c = str[i];
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
	}

    return hash;
}
#endif

template<typename T>
void checksum(T* data, int elemsize, int numelem, const char *name)
{
#ifdef QI_DBG_EXPORT
	uchar* cp = (uchar*)ALLOCA(elemsize*numelem*sizeof(T));
	hipDeviceSynchronize();
	hipMemcpy(cp, data, sizeof(T)*elemsize*numelem, hipMemcpyDeviceToHost);

	dbgprintf("%s:\n", name);
	for (int i=0;i<numelem;i++) {
		uchar *elem = cp+elemsize*sizeof(T)*i;
		dbgprintf("[%d]: %d\n", i, hash(elem, elemsize));
	}
#endif
}

template<typename TImageSampler>
void QueuedCUDATracker::ExecuteBatch(Stream *s)
{
	if (s->JobCount()==0)
		return;
	//dbgprintf("Sending %d images to GPU stream %p...\n", s->jobCount, s->stream);

	Device *d = s->device;
	hipSetDevice(d->index);

	BaseKernelParams kp;
	kp.d_imgmeans = s->d_imgmeans.data;
	kp.images = s->images;
	kp.njobs = s->JobCount();
	kp.d_params = s->d_locParams.data;

	hipEventRecord(s->batchStart, s->stream);
//	dbgprintf("copying %d jobs to gpu\n", s->JobCount());
	s->d_locParams.copyToDevice(s->locParams.data(), s->JobCount(), true, s->stream);

	{ScopedCPUProfiler p(&cpu_time.imageCopy);
		s->images.copyToDevice(s->hostImageBuf.data(), true, s->stream); 
	}
	//hipMemcpy2DAsync( s->images.data, s->images.pitch, s->hostImageBuf.data(), sizeof(float)*s->images.w, s->images.w*sizeof(float), s->images.h * s->JobCount(), hipMemcpyHostToDevice, s->stream); }
	//{ ProfileBlock p("jobs to gpu");
	//s->d_jobs.copyToDevice(s->jobs.data(), s->jobCount, true, s->stream); }

	if (!d->calib_gain.isEmpty()) {
		dim3 numThreads(16, 16, 2);
		dim3 numBlocks((cfg.width + numThreads.x - 1 ) / numThreads.x,
				(cfg.height + numThreads.y - 1) / numThreads.y,
				(s->JobCount() + numThreads.z - 1) / numThreads.z);

		gc_mutex.lock();
		float of = gc_offsetFactor, gf = gc_gainFactor;
		gc_mutex.unlock();

		ApplyOffsetGain <<< numBlocks, numThreads, 0, s->stream >>>	
			(s->JobCount(), s->images, s->d_locParams.data, s->device->calib_gain, s->device->calib_offset, gf, of);
	}

	hipEventRecord(s->imageCopyDone, s->stream);

	TImageSampler::BindTexture(s->images);
	{ ScopedCPUProfiler p(&cpu_time.com);
		BgCorrectedCOM<TImageSampler> <<< blocks(s->JobCount()), threads(), 0, s->stream >>> 
			(s->JobCount(), s->images, s->d_com.data, cfg.com_bgcorrection, s->d_imgmeans.data);
		checksum(s->d_com.data, 1, s->JobCount(), "com");
	}
	hipEventRecord(s->comDone, s->stream);

	device_vec<float3> *curpos = &s->d_com;
	if (s->localizeFlags & LT_QI) {
		ScopedCPUProfiler p(&cpu_time.qi);
		qi.Execute<TImageSampler> (kp, cfg, &s->qi_instance, &s->device->qi_instance, &s->d_com, &s->d_resultpos);
		curpos = &s->d_resultpos;
	}

	if (s->localizeFlags & LT_Gaussian2D) {
		G2MLE_Compute<TImageSampler> <<< blocks(s->JobCount()), threads(), 0, s->stream >>>
			(kp, cfg.gauss2D_sigma, cfg.gauss2D_iterations, s->d_com.data, s->d_resultpos.data, 0, 0);
		curpos = &s->d_resultpos;
	}

	hipEventRecord(s->qiDone, s->stream);

	{ScopedCPUProfiler p(&cpu_time.zcompute);

		zlutParams.img = d->radial_zlut;
		zlutParams.trigtable = d->zlut_trigtable.data;
		zlutParams.zcmpwindow = d->zcompareWindow.data;

		// Compute radial profiles
		if (s->localizeFlags & (LT_LocalizeZ | LT_BuildRadialZLUT)) {
			dim3 numThreads(16, 16);
			dim3 numBlocks( (s->JobCount() + numThreads.x - 1) / numThreads.x, 
					(cfg.zlut_radialsteps + numThreads.y - 1) / numThreads.y);
			ZLUT_RadialProfileKernel<TImageSampler> <<< numBlocks , numThreads, 0, s->stream >>>
				(s->JobCount(), s->images, zlutParams, curpos->data, s->d_radialprofiles.data, s->d_imgmeans.data);
			ZLUT_NormalizeProfiles<<< blocks(s->JobCount()), threads(), 0, s->stream >>> (s->JobCount(), zlutParams, s->d_radialprofiles.data);
		}
		// Store profile in LUT
		if (s->localizeFlags & LT_BuildRadialZLUT) {
			ZLUT_ProfilesToZLUT <<< blocks(s->JobCount()), threads(), 0, s->stream >>> (s->JobCount(), s->images, zlutParams, curpos->data, s->d_locParams.data, s->d_radialprofiles.data);
		}
		// Compute Z 
		if (s->localizeFlags & LT_LocalizeZ) {
			int zplanes = zlutParams.planes;
			dim3 numThreads(8, 16);
			ZLUT_ComputeProfileMatchScores <<< dim3( (s->JobCount() + numThreads.x - 1) / numThreads.x, (zplanes  + numThreads.y - 1) / numThreads.y), numThreads, 0, s->stream >>> 
				(s->JobCount(), zlutParams, s->d_radialprofiles.data, s->d_zlutcmpscores.data, s->d_locParams.data);
			ZLUT_ComputeZ <<< blocks(s->JobCount()), threads(), 0, s->stream >>> (s->JobCount(), zlutParams, curpos->data, s->d_zlutcmpscores.data, s->d_locParams.data);
		}
	}

	{ScopedCPUProfiler p(&cpu_time.imap);

		if (s->localizeFlags & LT_BuildImageLUT) {

		}

		if (s->localizeFlags & LT_IMAP) {
		}
	}

	TImageSampler::UnbindTexture(s->images);
	hipEventRecord(s->zcomputeDone, s->stream);

	{ ScopedCPUProfiler p(&cpu_time.getResults);
		s->d_com.copyToHost(s->com.data(), true, s->stream);
		curpos->copyToHost(s->results.data(), true, s->stream);
	}

	// Make sure we can query the all done signal
	hipEventRecord(s->localizationDone, s->stream);
}


void QueuedCUDATracker::CopyStreamResults(Stream *s)
{
	resultMutex.lock();
	for (int a=0;a<s->JobCount();a++) {
		LocalizationJob& j = s->jobs[a];
		LocalizationResult r;
		r.job = j;
		r.firstGuess =  vector2f( s->com[a].x, s->com[a].y );
		r.pos = vector3f( s->results[a].x , s->results[a].y, s->results[a].z);
		if(!(s->jobs[a].locType & LT_LocalizeZ))
			r.pos.z = 0.0f;

		results.push_back(r);
#ifdef _DEBUG
		dbgprintf("Bead: %d, Plane: %d, XYZ: %.4f, %.4f, %.4f\n", j.zlutIndex, j.zlutPlane, r.pos.x, r.pos.y, r.pos.z);
#endif
	}
	resultCount+=s->JobCount();
//	dbgprintf("Result count: %d\n", resultCount);
	resultMutex.unlock();

	// Update times
	float qi, com, imagecopy, zcomp, getResults;
	hipEventElapsedTime(&imagecopy, s->batchStart, s->imageCopyDone);
	hipEventElapsedTime(&com, s->imageCopyDone, s->comDone);
	hipEventElapsedTime(&qi, s->comDone, s->qiDone);
	hipEventElapsedTime(&zcomp, s->qiDone, s->zcomputeDone);
	hipEventElapsedTime(&getResults, s->zcomputeDone, s->localizationDone);
	time.com += com;
	time.qi += qi;
	time.imageCopy += imagecopy;
	time.zcompute += zcomp;
	time.getResults += getResults;
	batchesDone ++;
}

int QueuedCUDATracker::FetchResults(LocalizationResult* dstResults, int maxResults)
{
	resultMutex.lock();
	int numResults = 0;
	while (numResults < maxResults && !results.empty()) {
		dstResults[numResults++] = results.front();
		results.pop_front();
		resultCount--;
	}
	resultMutex.unlock();
	return numResults;
}

void QueuedCUDATracker::SetPixelCalibrationImages(float* offset, float* gain)
{
	for (uint i=0;i<devices.size();i++) {
		devices[i]->SetPixelCalibrationImages(offset, gain, cfg.width, cfg.height);
	}
}

void QueuedCUDATracker::SetPixelCalibrationFactors(float offsetFactor, float gainFactor)
{
	gc_mutex.lock();
	gc_gainFactor = gainFactor;
	gc_offsetFactor = offsetFactor;
	gc_mutex.unlock();
}

void QueuedCUDATracker::Device::SetPixelCalibrationImages(float* offset, float* gain, int img_width, int img_height)
{
	hipSetDevice(index);

	if (offset == 0) {
		calib_gain.free();
		calib_offset.free();
	}
	else if (radial_zlut.count > 0) {
		calib_gain = cudaImageListf::alloc(img_width,img_height,radial_zlut.count);
		calib_offset = cudaImageListf::alloc(img_width,img_height,radial_zlut.count);

		for (int j=0;j<radial_zlut.count;j++) {
			calib_gain.copyImageToDevice(j, &gain[img_width*img_height*j]);
			calib_offset.copyImageToDevice(j, &offset[img_width*img_height*j]);
		}
	}
}

// data can be zero to allocate ZLUT data
void QueuedCUDATracker::SetRadialZLUT(float* data,  int numLUTs, int planes, float* zcmp) 
{
	kernelParams.zlut.planes = planes;
	
	for (uint i=0;i<devices.size();i++) {
		devices[i]->SetRadialZLUT(data, cfg.zlut_radialsteps, planes, numLUTs, zcmp);
	}

	for (uint i=0;i<streams.size();i++) {
		StreamUpdateZLUTSize(streams[i]);
	}
}

void QueuedCUDATracker::StreamUpdateZLUTSize(Stream* s)
{		
	hipSetDevice(s->device->index);
	s->d_zlutcmpscores.init(s->device->radial_zlut.h * batchSize);
}

void QueuedCUDATracker::Device::SetRadialZLUT(float *data, int radialsteps, int planes, int numLUTs, float* zcmp)
{
	hipSetDevice(index);

	if (zcmp)
		zcompareWindow.copyToDevice(zcmp, radialsteps, false);
	else 
		zcompareWindow.free();

	radial_zlut = cudaImageListf::alloc(radialsteps, planes, numLUTs);
	if (data) {
		for (int i=0;i<numLUTs;i++)
			radial_zlut.copyImageToDevice(i, &data[planes*radialsteps*i]);
	}
	else radial_zlut.clear();
}

// delete[] memory afterwards
void QueuedCUDATracker::GetRadialZLUT(float* data)
{
	cudaImageListf* zlut = &devices[0]->radial_zlut;

	if (zlut->data) {
		for (int i=0;i<zlut->count;i++) {
			float* img = &data[i*cfg.zlut_radialsteps*zlut->h];
			zlut->copyImageToHost(i, img);
		}
	} else
		std::fill(data, data+(cfg.zlut_radialsteps*zlut->h*zlut->count), 0.0f);
}

void QueuedCUDATracker::GetRadialZLUTSize(int& count, int &planes, int& rsteps)
{
	count = devices[0]->radial_zlut.count;
	planes = devices[0]->radial_zlut.h;
	rsteps = cfg.zlut_radialsteps;
}


void QueuedCUDATracker::GetImageZLUTSize(int *dims)
{

}

void QueuedCUDATracker::GetImageZLUT(float* dst)
{

}

void QueuedCUDATracker::SetImageZLUT(float* dst,int* dims)
{
}


int QueuedCUDATracker::GetResultCount()
{
	resultMutex.lock();
	int r = resultCount;
	resultMutex.unlock();
	return r;
}

void QueuedCUDATracker::ClearResults()
{
	resultMutex.lock();
	results.clear();
	resultCount=0;
	resultMutex.unlock();
}


int QueuedCUDATracker::ScheduleFrame(uchar *imgptr, int pitch, int width, int height, ROIPosition *positions, int numROI, QTRK_PixelDataType pdt, const LocalizationJob* jobInfo)
{
	uchar* img = (uchar*)imgptr;
	int bpp = sizeof(float);
	if (pdt == QTrkU8) bpp = 1;
	else if (pdt == QTrkU16) bpp = 2;
	int count=0;
	for (int i=0;i<numROI;i++){
		ROIPosition pos = positions[i];
		if (pos.x < 0 || pos.y < 0 || pos.x + cfg.width > width || pos.y + cfg.height > height)
			continue;

		uchar *roiptr = &img[pitch * pos.y + pos.x * bpp];
		LocalizationJob job = *jobInfo;
		job.zlutIndex = i + jobInfo->zlutIndex;
		ScheduleLocalization(roiptr, pitch, pdt, &job);
		count ++;
	}
	return count;
}

std::string QueuedCUDATracker::GetProfileReport()
{
	float f = 1.0f/batchesDone;

	return deviceReport + "Time profiling: [GPU], [CPU] \n" +
		SPrintf("%d batches done of size %d, on %d streams", batchesDone, batchSize, streams.size()) + "\n" +
		SPrintf("Image copying: %.2f,\t%.2f ms\n", time.imageCopy*f, cpu_time.imageCopy*f) +
		SPrintf("QI:            %.2f,\t%.2f ms\n", time.qi*f, cpu_time.qi*f) +
		SPrintf("COM:           %.2f,\t%.2f ms\n", time.com*f, cpu_time.com*f) +
		SPrintf("Z Computing:   %.2f,\t%.2f ms\n", time.zcompute*f, cpu_time.zcompute*f);
}


QueuedCUDATracker::ConfigValueMap QueuedCUDATracker::GetConfigValues()
{
	ConfigValueMap cvm;
	cvm["use_texturecache"] = useTextureCache ? "1" : "0";
	return cvm;
}

void QueuedCUDATracker::SetConfigValue(std::string name, std::string value)
{
	if (name == "use_texturecache")
		useTextureCache = atoi(value.c_str()) != 0;
}



