#include "hip/hip_runtime.h"
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include "std_incl.h"
#include "hip/hip_vector_types.h"

#include "QueuedCUDATracker.h"
#include "gpu_utils.h"
#include "simplefft.h"

#include "utils.h"

#define LSQFIT_FUNC __device__ __host__
#include "LsqQuadraticFit.h"

// Types used by QI algorithm
typedef float qivalue_t;
typedef sfft::complex<qivalue_t> qicomplex_t;

// This template specialization makes sure that we dont link against cudaSharedMemory from host-side code (You get a linker error if you do)
__shared__ float2 cudaSharedMemory[];
template<bool cpuMode> struct shared_mem {};
template<> struct shared_mem<true> {
	CUBOTH static float2* sharedMemory(float2* sharedBuf) { return sharedBuf; }
};
template<> struct shared_mem<false> {
	CUBOTH static float2* sharedMemory(float2* sharedBuf) { return cudaSharedMemory; }
};

// QueuedCUDATracker allows runtime choosing of GPU or CPU code. All GPU kernel calls are done through the following macro:
// Depending on 'useCPU' it either invokes a CUDA kernel named 'Funcname', or simply loops over the data on the CPU side calling 'Funcname' for each image
#define KERNEL_DISPATCH(Funcname, TParam) \
__global__ void Funcname##Kernel(cudaImageListf images, TParam param) { \
	int idx = blockIdx.x * blockDim.x + threadIdx.x; \
	if (idx < images.count) { \
		Funcname<false>(idx, images, param); \
	} \
} \
void QueuedCUDATracker::CallKernel_##Funcname(cudaImageListf& images, TParam param, uint sharedMem)  { \
	if (useCPU) { \
		for (int idx=0;idx<images.count;idx++) { \
			::Funcname <true> (idx, images, param); \
		} \
	} else { \
		Funcname##Kernel <<<blocks(images.count), threads(), sharedMem>>> (images,param); \
	} \
}

QueuedTracker* CreateQueuedTracker(QTrkSettings* cfg)
{
	return new QueuedCUDATracker(cfg);
}

QueuedCUDATracker::QueuedCUDATracker(QTrkSettings *cfg)
{
	this->cfg = *cfg;

	hipGetDeviceProperties(&deviceProp, 0);

	batchSize = numThreads * deviceProp.multiProcessorCount;

//	forward_fft = new cudafft<float>(cfg->xc1_profileLength, false);
//	backward_fft = new cudafft<float>(cfg->xc1_profileLength, true);

	//int sharedSpacePerThread = (prop.sharedMemPerBlock-forward_fft->kparams_size*2) / numThreads;
//	dbgprintf("2X FFT instance requires %d bytes. Space per thread: %d\n", forward_fft->kparams_size*2, sharedSpacePerThread);
	dbgprintf("Device: %s\n", deviceProp.name);
	dbgprintf("Shared memory space:%d bytes. Per thread: %d\n", deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerBlock/numThreads);
	dbgprintf("# of CUDA processors:%d\n", deviceProp.multiProcessorCount);
	dbgprintf("warp size: %d\n", deviceProp.warpSize);

	useCPU = false;//cfg->numThreads == 0;
	qiProfileLen = 1;
	while (qiProfileLen < cfg->qi_radialsteps) qiProfileLen *= 2;

	fft_twiddles = DeviceMem( sfft::fill_twiddles<float> (qiProfileLen) );
	sharedBuf.init(qiProfileLen*2*batchSize);
	sharedMemSize = sizeof(float2) * cfg->qi_radialsteps*2*numThreads; 
	KernelParams &p = kernelParams;
	QIParams qi = p.qi_params;
	qi.angularSteps = cfg->qi_angularsteps;
	qi.iterations = cfg->qi_iterations;
	qi.maxRadius = cfg->qi_maxradius;
	qi.minRadius = cfg->qi_minradius;
	qi.radialSteps = cfg->qi_radialsteps;
	qi.d_twiddles = fft_twiddles.data;
	p.sharedBuf = sharedBuf.data;
	p.useShared = qi.radialSteps*2 * numThreads < deviceProp.sharedMemPerBlock;

	dbgprintf("Required shared memory: %d\n", sharedMemSize);

	currentBatch = AllocBatch();
}

QueuedCUDATracker::~QueuedCUDATracker()
{
	DeleteAllElems(freeBatches);
	DeleteAllElems(active);
}

void QueuedCUDATracker::GenerateTestImage(float* dst, float xp,float yp, float z, float photoncount)
{
}



template<bool cpuMode>
static CUBOTH float2 BgCorrectedCOM(int idx, cudaImageListf& images)
{
	int imgsize = images.w*images.h;
	float sum=0, sum2=0;
	float momentX=0;
	float momentY=0;

	for (int y=0;y<images.h;y++)
		for (int x=0;x<images.w;x++) {
			float v = images.pixel(x,y,idx);
			sum += v;
			sum2 += v*v;
		}

	float invN = 1.0f/imgsize;
	float mean = sum * invN;
	float stdev = sqrtf(sum2 * invN - mean * mean);
	sum = 0.0f;

	for (int y=0;y<images.h;y++)
		for(int x=0;x<images.w;x++)
		{
			float v = images.pixel(x,y,idx);
			v = fabs(v-mean)-2.0f*stdev;
			if(v<0.0f) v=0.0f;
			sum += v;
			momentX += x*v;
			momentY += y*v;
		}

	float2 com;
	com.x = momentX / (float)sum;
	com.y = momentY / (float)sum;
	return com;
}


__global__ void BgCorrectedCOMKernel(cudaImageListf images, float2* d_com) { 
	int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (idx < images.count) { 
		d_com[idx] = BgCorrectedCOM<false>(idx, images);
	} 
} 


void QueuedCUDATracker::ComputeBgCorrectedCOM(cudaImageListf& images, float2* d_com)
{
	BgCorrectedCOMKernel<<<blocks(images.count), threads()>>> (images, d_com);
}


template<bool cpuMode>
static CUBOTH void MakeTestImage(int idx, cudaImageListf& images, float3* d_positions)
{
	float3 pos = d_positions[idx];
	
	float S = 1.0f/pos.z;
	for (int y=0;y<images.h;y++) {
		for (int x=0;x<images.w;x++) {
			float X = x - pos.x;
			float Y = y - pos.y;
			float r = sqrtf(X*X+Y*Y)+1;
			float v = sinf(r/(5*S)) * expf(-r*r*S*0.01f);
			images.pixel(x,y,idx) = v;
		}
	}
}


KERNEL_DISPATCH(MakeTestImage, float3*); 

void QueuedCUDATracker::GenerateImages(cudaImageListf& imgList, float3* d_pos)
{
	CallKernel_MakeTestImage(imgList, d_pos);
}


texture<float, hipTextureType2D, hipReadModeElementType> qi_image_texture(0, hipFilterModeLinear);


template<bool cpuMode>
static CUBOTH qivalue_t QI_ComputeOffset(qicomplex_t* profile, qicomplex_t* tmpbuf, const QIParams& params, int idx) {
	int nr = params.radialSteps;

	qicomplex_t* reverse;
	reverse = tmpbuf;

	for(int x=0;x<nr*2;x++)
		reverse[x] = profile[nr*2-1-x];

	sfft::fft_forward(nr*2, profile, params.d_twiddles);
	sfft::fft_forward(nr*2, reverse, params.d_twiddles);

	// multiply with conjugate
	for(int x=0;x<nr*2;x++)
		profile[x] = profile[x] * reverse[x].conjugate();

	sfft::fft_inverse(nr*2, profile, params.d_twiddles);
	// fft_out2 now contains the autoconvolution
	// convert it to float
	qivalue_t* autoconv = (qivalue_t*)tmpbuf;
	for(int x=0;x<nr*2;x++)  {
		autoconv[x] = profile[(x+nr)%(nr*2)].real();
	}

	float maxPos = ComputeMaxInterp<qivalue_t, 5>(autoconv, nr*2);
	return (maxPos - nr) / (3.14159265359f * 0.5f);
}


static CUBOTH void ComputeQuadrantProfile(cudaImageListf& images, int idx, float* dst, const QIParams& params, int quadrant, float2 center)
{
	const int qmat[] = {
		1, 1,
		-1, 1,
		-1, -1,
		1, -1 };
	int mx = qmat[2*quadrant+0];
	int my = qmat[2*quadrant+1];

	for (int i=0;i<params.radialSteps;i++)
		dst[i]=0.0f;
	
	double total = 0.0f;
	float rstep = (params.maxRadius - params.minRadius) / params.radialSteps;
	for (int i=0;i<params.radialSteps; i++) {
		double sum = 0.0f;
		float r = params.minRadius + rstep * i;

		for (int a=0;a<params.angularSteps;a++) {
			float ang = 0.5f*3.141593f*a/(float)params.angularSteps;
			float x = center.x + mx*cos(ang) * r;
			float y = center.y + my*sin(ang) * r;
			sum += images.interpolate(x, y, idx);
		}

		dst[i] = sum/params.angularSteps-images.borderValue;
		total += dst[i];
	}
}

template<bool cpuMode>
static CUBOTH void ComputeQI(int idx, cudaImageListf& images, KernelParams params, float3* d_initial, float2* d_output, uint* d_boundaryHits)
{
	QIParams& qp = params.qi_params;
	int nr=qp.radialSteps;
	float2 center = make_float2(d_initial[idx].x, d_initial[idx].y);

	float pixelsPerProfLen = (qp.maxRadius-qp.minRadius)/qp.radialSteps;
	bool boundaryHit = false;

	size_t total_required = sizeof(qivalue_t)*nr*4 + sizeof(qicomplex_t)*nr*2;

	qivalue_t* buf = (qivalue_t*)malloc(total_required);
	qivalue_t* q0=buf, *q1=buf+nr, *q2=buf+nr*2, *q3=buf+nr*3;

	qicomplex_t* concat0 = (qicomplex_t*)(buf + nr*4);
	qicomplex_t* concat1 = concat0 + nr;
	qicomplex_t* tmpbuf = (qicomplex_t*) &shared_mem<cpuMode>::sharedMemory(params.sharedBuf) [idx * nr*2];
	for (int k=0;k<qp.iterations;k++){
		// check bounds
		boundaryHit = images.boundaryHit(center, qp.maxRadius);

		for (int q=0;q<4;q++) {
			ComputeQuadrantProfile(images, idx, buf+q*nr, qp, q, center);
		}
		
		// Build Ix = qL(-r) || qR(r)
		// qL = q1 + q2   (concat0)
		// qR = q0 + q3   (concat1)
		for(int r=0;r<nr;r++) {
			concat0[nr-r-1] = qicomplex_t(q1[r]+q2[r]);
			concat1[r] = qicomplex_t(q0[r]+q3[r]);
		}

		float offsetX = QI_ComputeOffset<cpuMode>(concat0, tmpbuf, qp, idx);

		// Build Iy = qB(-r) || qT(r)
		// qT = q0 + q1
		// qB = q2 + q3
		for(int r=0;r<nr;r++) {
			concat0[r] = qicomplex_t(q0[r]+q1[r]);
			concat1[nr-r-1] = qicomplex_t(q2[r]+q3[r]);
		}
		float offsetY = QI_ComputeOffset<cpuMode>(concat0, tmpbuf, qp, idx);

		//printf("[%d] OffsetX: %f, OffsetY: %f\n", k, offsetX, offsetY);
		center.x += offsetX * pixelsPerProfLen;
		center.y += offsetY * pixelsPerProfLen;
	}

	d_output[idx] = center;
	if (d_boundaryHits) d_boundaryHits[idx] = boundaryHit;

	free(buf);
}

/*
__global__ void ComputeQIKernel(cudaImageListf images, QIParams param) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (idx < images.count) { 
		ComputeQI<false>(idx, images, param);
	} 
}
*/
void QueuedCUDATracker::ComputeQI(cudaImageListf& images, float2* d_initial, float2* d_result)
{

	if (!useCPU) {
		images.bind(qi_image_texture);
	}

	//ComputeQIKernel <<<blocks(images.count), threads(), sharedMemSize>>> (images, params);

	if (!useCPU)
		images.unbind(qi_image_texture);
}

QueuedCUDATracker::Batch::~Batch() 
{
	if(images.data) images.free();
	hipHostFree(hostImageBuf);
	hipEventDestroy(localizationDone);
	hipEventDestroy(imageBufferCopied);
}

QueuedCUDATracker::Batch* QueuedCUDATracker::AllocBatch()
{
	if (freeBatches.empty()) { // allocate more batches?
		Batch* b = new Batch();
		
		uint hostBufSize = sizeof(float)* cfg.width*cfg.height*batchSize;
		hipHostMalloc(&b->hostImageBuf, hostBufSize, hipHostMallocWriteCombined);
		b->images = cudaImageListf::alloc(cfg.width,cfg.height,batchSize, useCPU);
		b->d_jobs.init(batchSize);
		hipEventCreate(&b->localizationDone);
		hipEventCreate(&b->imageBufferCopied);

		return b;
	} else {
		Batch* batch = freeBatches.back();
		freeBatches.pop_back();
		return batch;
	}
}



void QueuedCUDATracker::Start() 
{

}


void QueuedCUDATracker::ClearResults()
{
	FetchResults();
	results.clear();
}

__global__ void LocalizeBatchKernel(int numImages, cudaImageListf images, KernelParams params, CUDATrackerJob* jobs)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx >= numImages)
		return;

	float2 com = BgCorrectedCOM<false>(idx, images);
	jobs[idx].firstGuess = com;
	jobs[idx].resultPos.x = com.x;
	jobs[idx].resultPos.y = com.y;
}

void QueuedCUDATracker::ScheduleLocalization(uchar* data, int pitch, QTRK_PixelDataType pdt, LocalizeType locType, uint id, vector3f* initialPos, uint zlutIndex, uint zlutPlane)
{
	CUDATrackerJob job;
	if (initialPos) 
		job.initialPos = *(float3*)initialPos;
	job.id = id;
	job.zlut = zlutIndex;
	job.locType = locType;

	Batch* cb = currentBatch;
	cb->jobs.push_back(job);

	// Copy the image to the batch image buffer (CPU side)
	float* hostbuf = &cb->hostImageBuf[cfg.height*cfg.width* (cb->jobs.size()-1)];
	uchar* srcptr = data;
	CopyImageToFloat(data, cfg.width, cfg.height, pitch, pdt, hostbuf);

	// If batch is filled, copy the image to video memory asynchronously, and start the localization
	if (cb->jobs.size() == batchSize)
		QueueCurrentBatch();
}


void QueuedCUDATracker::QueueCurrentBatch()
{
	Batch* cb = currentBatch;
	hipMemcpy2DAsync(cb->images.data, cb->images.pitch, cb->hostImageBuf, 
		sizeof(float)*cfg.width, cfg.width*sizeof(float), cfg.height*cb->jobs.size(), hipMemcpyHostToDevice);
	hipMemcpyAsync(cb->d_jobs.data, &cb->jobs[0], sizeof(CUDATrackerJob) * cb->jobs.size(), hipMemcpyHostToDevice);

	hipEventRecord(cb->imageBufferCopied);
	LocalizeBatchKernel<<<blocks(cb->jobs.size()), threads(), sharedMemSize>>> (cb->jobs.size(), cb->images, kernelParams, cb->d_jobs.data);
	// Copy back the results
	hipMemcpyAsync(&cb->jobs[0], cb->d_jobs.data, sizeof(CUDATrackerJob) * cb->jobs.size(), hipMemcpyDeviceToHost);

	// Make sure we can query the all done signal
	hipEventRecord(currentBatch->localizationDone);

	active.push_back(currentBatch);
	currentBatch = AllocBatch();
}

void QueuedCUDATracker::Flush()
{
	QueueCurrentBatch();
}

void QueuedCUDATracker::FetchResults()
{
	auto i = active.begin();
	
	while (i != active.end())
	{
		auto cur = i++;
		Batch* b = *cur;

		hipError_t result = hipEventQuery(b->localizationDone);
		if (result == hipSuccess) {
			CopyBatchResults(b);
			active.erase(cur);
			freeBatches.push_back(b);
		}
	}
}

void QueuedCUDATracker::CopyBatchResults(Batch *b)
{
	for (int a=0;a<b->jobs.size();a++) {
		auto j = b->jobs[a];

		LocalizationResult r;
		r.error = j.error;
		r.id = j.id;
		r.firstGuess.x = j.firstGuess.x; r.firstGuess.y = j.firstGuess.y;
		r.locType = j.locType;
		r.zlutIndex = j.zlut;
		r.pos.x = j.resultPos.x;
		r.pos.y = j.resultPos.y;
		r.z = j.resultPos.z;

		results.push_back(r);
	}

	b->jobs.clear();
}

int QueuedCUDATracker::PollFinished(LocalizationResult* dstResults, int maxResults)
{
	FetchResults();
	int numResults = 0;
	while (numResults < maxResults && !results.empty()) {
		dstResults[numResults++] = results.back();
		results.pop_back();
	}
	return numResults;
}

// data can be zero to allocate ZLUT data
void QueuedCUDATracker::SetZLUT(float* data,  int numLUTs, int planes, int res) 
{
	zlut_planes = planes;
	zlut_count = numLUTs;
	zlut_res = res;
	if (data) zlut.copyFrom(data, false);
}

// delete[] memory afterwards
float* QueuedCUDATracker::GetZLUT(int *count, int* planes, int *res)
{
	float* data = new float[zlut_planes * zlut_res * zlut_count];
	zlut.copyTo(data, false);
	if (planes) *planes = zlut_planes;
	if (res) *res = zlut_res;
	if (count) *count = zlut_count;
	return data;
}




int QueuedCUDATracker::GetResultCount()
{
	FetchResults();
	return results.size();
}

