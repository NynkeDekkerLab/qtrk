#include "hip/hip_runtime.h"
/*
Quadrant Interpolation on CUDA

Method:

-Batch images into host-side buffer
-Running batch:
	- Async copy host-side buffer to device
	- Bind image
	- Run COM kernel
	- QI loop: {
		- Run QI kernel: Sample from texture into quadrant profiles
		- Run CUFFT. Each iteration per axis does 2x forward FFT, and 1x backward FFT.
		- Run QI kernel: Compute positions
	}
	- Compute ZLUT profiles
	- Depending on localize flags:
		- copy ZLUT profiles (for ComputeBuildZLUT flag)
		- generate compare profile kernel + compute Z kernel (for ComputeZ flag)
	- Unbind image
	- Async copy results to host


Issues:
- Due to FPU operations on texture coordinates, there are small numerical differences between localizations of the same image at a different position in the batch
- 
*/
#include "std_incl.h"
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_vector_types.h"
#include <cstdint>
#include "utils.h"

#include "QueuedCUDATracker.h"
#include "gpu_utils.h"
#include "ImageSampler.h"

#define LSQFIT_FUNC __device__ __host__
#include "LsqQuadraticFit.h"

#include "Kernels.h"


// Do CPU-side profiling of kernel launches?
//#define TRK_PROFILE

#ifdef TRK_PROFILE
	class ProfileBlock
	{
		double start;
		const char *name;

	public:
		typedef std::pair<int, double> Item;
		static std::map<const char*, Item> results;

		ProfileBlock(const char* name) : name (name) {
			start = GetPreciseTime();
		}
		~ProfileBlock() {
			double end = GetPreciseTime();
			//dbgprintf("%s took %.2f ms\n", name, (end-start)*1000.0f);
			if (results.find(name) == results.end())
				results[name] = Item(1, end-start);
			else {
				Item prev = results[name];
				results[name] = Item (prev.first+1, end-start + prev.second);
			}
		}
	};
	QueuedCUDATracker::ProfileResults ProfileBlock::results;
	QueuedCUDATracker::ProfileResults QueuedCUDATracker::GetProfilingResults() { return ProfileBlock::results; };
#else
	class ProfileBlock {
	public:
		ProfileBlock(const char *name) {}
	};
	QueuedCUDATracker::ProfileResults QueuedCUDATracker::GetProfilingResults() { return QueuedCUDATracker::ProfileResults(); };
#endif

static std::vector<int> cudaDeviceList; 
void SetCUDADevices(int* dev, int numdev) {
	cudaDeviceList.assign(dev,dev+numdev);
}



QueuedTracker* CreateQueuedTracker(QTrkSettings* cfg)
{
	return new QueuedCUDATracker(cfg);
}

void CheckCUDAError()
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		const char* errstr = hipGetErrorString(err);
		dbgprintf("CUDA error: %s\n" ,errstr);
	}
}

static int GetBestCUDADevice()
{
	int bestScore;
	int bestDev;
	int numDev;
	hipGetDeviceCount(&numDev);
	for (int a=0;a<numDev;a++) {
		int score;
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, a);
		score = prop.multiProcessorCount * prop.clockRate;
		if (a==0 || bestScore < score) {
			bestScore = score;
			bestDev = a;
		}
	}
	return bestDev;
}

void QueuedCUDATracker::InitializeDeviceList()
{
	int numDevices;
	hipGetDeviceCount(&numDevices);

	// Select the most powerful one
	if (cfg.cuda_device == QTrkCUDA_UseBest) {
		cfg.cuda_device = GetBestCUDADevice();
		devices.push_back(new Device(cfg.cuda_device));
	} else if(cfg.cuda_device == QTrkCUDA_UseAll) {
		// Use all devices
		for (int i=0;i<numDevices;i++)
			devices.push_back(new Device(i));
	} else if (cfg.cuda_device == QTrkCUDA_UseList) {
		for (uint i=0;i<cudaDeviceList.size();i++)
			devices.push_back(new Device(cudaDeviceList[i]));
	} else {
		devices.push_back (new Device(cfg.cuda_device));
	}
	dbgprintf("Using devices: ");
	for (uint i=0;i<devices.size();i++) {
		hipDeviceProp_t p; 
		hipGetDeviceProperties(&p, devices[i]->index);
		dbgprintf("%s%s", p.name, i<devices.size()-1?", ":"\n");
	}
}


QueuedCUDATracker::QueuedCUDATracker(QTrkSettings *cfg, int batchSize)
{
	this->cfg = *cfg;

	InitializeDeviceList();

	// We take numThreads to be the number of CUDA streams
	if (cfg->numThreads < 1) {
		cfg->numThreads = devices.size()*3;
	}
	int numStreams = cfg->numThreads;

	hipGetDeviceProperties(&deviceProp, devices[0]->index);
	numThreads = deviceProp.warpSize;
	
	if(batchSize<0) batchSize = 512;
	while (batchSize * cfg->height > deviceProp.maxTexture2D[1]) {
		batchSize/=2;
	}
	this->batchSize = batchSize;

	qi_FFT_length = cfg->qi_radialsteps*2;

	dbgprintf("# of CUDA processors:%d. Using %d streams\n", deviceProp.multiProcessorCount, numStreams);
	dbgprintf("Warp size: %d. Max threads: %d, Batch size: %d. QI FFT Length: %d\n", deviceProp.warpSize, deviceProp.maxThreadsPerBlock, batchSize, qi_FFT_length);

	KernelParams &p = kernelParams;
	p.com_bgcorrection = cfg->com_bgcorrection;
	
	ZLUTParams& zp = p.zlut;
	zp.angularSteps = cfg->zlut_angularsteps;
	zp.maxRadius = cfg->zlut_maxradius;
	zp.minRadius = cfg->zlut_minradius;
	zp.planes = 0;
	zp.zcmpwindow = 0;

	QIParams& qi = p.qi;
	qi.angularSteps = cfg->qi_angsteps_per_quadrant;
	qi.iterations = cfg->qi_iterations;
	qi.maxRadius = cfg->qi_maxradius;
	qi.minRadius = cfg->qi_minradius;
	qi.radialSteps = cfg->qi_radialsteps;
	std::vector<float2> qi_radialgrid(qi.angularSteps);
	for (int i=0;i<qi.angularSteps;i++)  {
		float ang = 0.5f*3.141593f*i/(float)qi.angularSteps;
		qi_radialgrid[i]=make_float2(cos(ang), sin(ang));
	}

	std::vector<float2> zlut_radialgrid(cfg->zlut_angularsteps);
	for (int i=0;i<cfg->zlut_angularsteps;i++) {
		float ang = 2*3.141593f*i/(float)cfg->zlut_angularsteps;
		zlut_radialgrid[i]=make_float2(cos(ang),sin(ang));
	}

	for (uint i=0;i<devices.size();i++) {
		Device* d = devices[i];
		hipSetDevice(d->index);
		d->d_qiradialgrid=qi_radialgrid;
		d->d_zlutradialgrid = zlut_radialgrid;
	}
	kernelParams.zlut.img = cudaImageListf::emptyList();
	
	streams.reserve(numStreams);
	try {
		for (int i=0;i<numStreams;i++)
			streams.push_back( CreateStream( devices[i%devices.size()] ) );
	}
	catch(...) {
		DeleteAllElems(streams);
		throw;
	}

	currentStream=streams[0];
	streams[0]->OutputMemoryUse();

	batchesDone = 0;
	time_QI = time_COM = time_ZCompute = time_imageCopy = 0.0;
	useTextureCache = false;
}

QueuedCUDATracker::~QueuedCUDATracker()
{
	DeleteAllElems(streams);
	DeleteAllElems(devices);	
}

QueuedCUDATracker::Device::~Device()
{
	hipSetDevice(index);
	zlut.free();
}


QueuedCUDATracker::Stream::Stream()
{ 
	device = 0;
	hostImageBuf = 0; 
	images.data=0; 
	stream=0;
	state = StreamIdle;
	localizeFlags=0;
}

QueuedCUDATracker::Stream::~Stream() 
{
	hipSetDevice(device->index);
	hipfftDestroy(fftPlan);

	if(images.data) images.free();
	hipEventDestroy(localizationDone);
	hipEventDestroy(qiDone);
	hipEventDestroy(comDone);
	hipEventDestroy(imageCopyDone);
	hipEventDestroy(zcomputeDone);
	hipEventDestroy(batchStart);

	if (stream)
		hipStreamDestroy(stream); // stream can be zero if in debugStream mode.
}



bool QueuedCUDATracker::Stream::IsExecutionDone()
{
	hipSetDevice(device->index);
	return hipEventQuery(localizationDone) == hipSuccess;
}


void QueuedCUDATracker::Stream::OutputMemoryUse()
{
	int deviceMem = d_com.memsize() + d_zlutmapping.memsize() + d_QIprofiles.memsize() + d_QIprofiles_reverse.memsize() + d_radialprofiles.memsize() + d_imgmeans.memsize() +
		d_quadrants.memsize() + d_resultpos.memsize() + d_zlutcmpscores.memsize() + images.totalNumBytes();

	int hostMem = hostImageBuf.memsize() + com.memsize() + zlutmapping.memsize() + results.memsize();

	dbgprintf("Stream memory use: %d kb pinned on host, %d kb device memory (%d for images). \n", hostMem / 1024, deviceMem/1024, images.totalNumBytes()/1024);
}


QueuedCUDATracker::Stream* QueuedCUDATracker::CreateStream(Device* device)
{
	Stream* s = new Stream();

	try {
		s->device = device;
		hipSetDevice(device->index);
		hipStreamCreate(&s->stream);

		s->images = cudaImageListf::alloc(cfg.width, cfg.height, batchSize);
		s->images.allocateHostImageBuffer(s->hostImageBuf);

		s->jobs.reserve(batchSize);
		s->results.init(batchSize);
		s->com.init(batchSize);
		s->d_com.init(batchSize);
		s->d_resultpos.init(batchSize);
		s->results.init(batchSize);
		s->zlutmapping.init(batchSize);
		s->d_zlutmapping.init(batchSize);
		s->d_quadrants.init(qi_FFT_length*batchSize*2);
		s->d_QIprofiles.init(batchSize*2*qi_FFT_length); // (2 axis) * (2 radialsteps) = 8 * nr = 2 * qi_FFT_length
		s->d_QIprofiles_reverse.init(batchSize*2*qi_FFT_length);
		s->d_radialprofiles.init(cfg.zlut_radialsteps*batchSize);
		s->d_imgmeans.init(batchSize);
		s->d_shiftbuffer.init(qi_FFT_length * batchSize);
		
		// 2* batchSize, since X & Y both need an FFT transform
		//hipfftResult_t r = hipfftPlanMany(&s->fftPlan, 1, &qi_FFT_length, 0, 1, qi_FFT_length, 0, 1, qi_FFT_length, HIPFFT_C2C, batchSize*4);
		hipfftResult_t r = hipfftPlan1d(&s->fftPlan, qi_FFT_length, HIPFFT_C2C, batchSize*2);

		if(r != HIPFFT_SUCCESS) {
			throw std::runtime_error( SPrintf("CUFFT plan creation failed. FFT len: %d. Batchsize: %d\n", qi_FFT_length, batchSize*4));
		}
		cufftSetCompatibilityMode(s->fftPlan, CUFFT_COMPATIBILITY_NATIVE);
		hipfftSetStream(s->fftPlan, s->stream);

		hipEventCreate(&s->localizationDone);
		hipEventCreate(&s->comDone);
		hipEventCreate(&s->imageCopyDone);
		hipEventCreate(&s->zcomputeDone);
		hipEventCreate(&s->qiDone);
		hipEventCreate(&s->batchStart);
	} catch (...) {
		delete s;
		throw;
	}
	return s;
}

 // get a stream that not currently executing, and still has room for images
QueuedCUDATracker::Stream* QueuedCUDATracker::GetReadyStream()
{
	if (currentStream && currentStream->state != Stream::StreamExecuting && 
		currentStream->jobs.size() < batchSize) {
		return currentStream;
	}

	// Find another stream that is ready
	// First round: Check streams with current non-updated state. 
	// Second round: Query the GPU again for updated stream state.
	// Further rounds: Wait 1 ms and try again.
	for (int i = 0; true; i ++) {
		for (uint a=0;a<streams.size();a++) {
			Stream *s = streams[a];
			if (s->state != Stream::StreamExecuting) {
				currentStream = s;
				dbgprintf("Switching to stream %d\n", a);
				return s;
			}
		}
		FetchResults();
		if (i > 0) Threads::Sleep(1);
	}
}





void QueuedCUDATracker::ClearResults()
{
	FetchResults();
	results.clear();
}

// All streams on StreamIdle?
bool QueuedCUDATracker::IsIdle()
{
	return CheckAllStreams(Stream::StreamIdle) && (!currentStream || currentStream->jobs.empty() );
}

bool QueuedCUDATracker::CheckAllStreams(Stream::State s)
{
	FetchResults();
	for (uint a=0;a<streams.size();a++){
		if (streams[a]->state != s)
			return false;
	}
	return true;
}

bool QueuedCUDATracker::IsQueueFilled()
{
	return CheckAllStreams(Stream::StreamExecuting);
}

void QueuedCUDATracker::ScheduleLocalization(uchar* data, int pitch, QTRK_PixelDataType pdt, const LocalizationJob* jobInfo )
{
	Stream* s = GetReadyStream();
	s->lock();

	int jobIndex = s->jobs.size();
	LocalizationJob job = *jobInfo;
	job.locType = jobInfo->LocType();
	if (s->device->zlut.isEmpty())  // dont do ZLUT commands when no ZLUT has been set
		job.locType &= ~(LocalizeZ | LocalizeBuildZLUT);
	s->jobs.push_back(job);
	s->localizeFlags |= job.locType; // which kernels to run
	s->zlutmapping[jobIndex].locType = job.LocType();
	s->zlutmapping[jobIndex].zlutIndex = jobInfo->zlutIndex;
	s->zlutmapping[jobIndex].zlutPlane = jobInfo->zlutPlane;

	// Copy the image to the batch image buffer (CPU side)
	float* hostbuf = &s->hostImageBuf[cfg.height*cfg.width*jobIndex];
	CopyImageToFloat(data, cfg.width, cfg.height, pitch, pdt, hostbuf);

//	tmp = floatToNormalizedInt( (float*)hostbuf, cfg.width,cfg.height,(uchar)255);
//	WriteJPEGFile(tmp, cfg.width,cfg.height, "writehostbuf2.jpg", 99);
//	delete[] tmp;

	// If batch is filled, copy the image to video memory asynchronously, and start the localization
	if (s->jobs.size() == batchSize) {
		if (useTextureCache)
			ExecuteBatch<ImageSampler_Tex> (s);
		else
			ExecuteBatch<ImageSampler_MemCopy> (s);
	}

	s->unlock();
}


void QueuedCUDATracker::Flush()
{
	if (currentStream && currentStream->state == Stream::StreamIdle) {
		currentStream->lock();

		if (useTextureCache) 
			ExecuteBatch<ImageSampler_Tex> (currentStream);
		else 
			ExecuteBatch<ImageSampler_MemCopy> (currentStream);

		currentStream->unlock();
		currentStream = 0;
	}
}


#ifdef QI_DBG_EXPORT
static unsigned long hash(unsigned char *str, int n)
{
    unsigned long hash = 5381;
    
    for (int i=0;i<n;i++) {
		int c = str[i];
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
	}

    return hash;
}
#endif

template<typename T>
void checksum(T* data, int elemsize, int numelem, const char *name)
{
#ifdef QI_DBG_EXPORT
	uchar* cp = (uchar*)ALLOCA(elemsize*numelem*sizeof(T));
	hipDeviceSynchronize();
	hipMemcpy(cp, data, sizeof(T)*elemsize*numelem, hipMemcpyDeviceToHost);

	dbgprintf("%s:\n", name);
	for (int i=0;i<numelem;i++) {
		uchar *elem = cp+elemsize*sizeof(T)*i;
		dbgprintf("[%d]: %d\n", i, hash(elem, elemsize));
	}
#endif
}

template<typename TImageSampler>
void QueuedCUDATracker::QI_Iterate(device_vec<float3>* initial, device_vec<float3>* newpos, Stream *s)
{
	int njobs = s->jobs.size();
	dim3 qdrThreads(16, 8);


	if (0) {
		dim3 qdrDim( (njobs + qdrThreads.x - 1) / qdrThreads.x, (cfg.qi_radialsteps + qdrThreads.y - 1) / qdrThreads.y, 4 );
		QI_ComputeQuadrants<TImageSampler> <<< qdrDim , qdrThreads, 0, s->stream >>> 
			(njobs, s->images, initial->data, s->d_quadrants.data, s->d_imgmeans.data, kernelParams.qi);

		QI_QuadrantsToProfiles <<< blocks(njobs), threads(), 0, s->stream >>> 
			(njobs, s->images, s->d_quadrants.data, s->d_QIprofiles.data, s->d_QIprofiles_reverse.data, kernelParams.qi);
	}
	else {
		QI_ComputeProfile <TImageSampler> <<< blocks(njobs), threads(), 0, s->stream >>> (njobs, s->images, initial->data, 
			s->d_quadrants.data, s->d_QIprofiles.data, s->d_QIprofiles_reverse.data, s->d_imgmeans.data,  kernelParams.qi);
	}
	/*
	hipStreamSynchronize(s->stream);
	auto q0 = s->d_quadrants.toVector();
	auto p0 = s->d_QIprofiles.toVector();

	WriteImageAsCSV("qi-qtc.txt", &q0[0], cfg.qi_radialsteps * 4, njobs);
	WriteComplexImageAsCSV("qi-ptc.txt", (std::complex<float>*)&p0[0], 2*qi_FFT_length, njobs);

	QI_ComputeProfile <TImageSampler> <<< blocks(njobs), threads(), 0, s->stream >>> (njobs, s->images, initial->data, 
		s->d_quadrants.data, s->d_QIprofiles.data, s->d_QIprofiles_reverse.data, s->d_imgmeans.data,  kernelParams.qi);
	hipStreamSynchronize(s->stream);
	auto q1 = s->d_quadrants.toVector();
	auto p1 = s->d_QIprofiles.toVector();

	WriteImageAsCSV("qi-q1.txt", &q1[0], cfg.qi_radialsteps * 4, njobs);
	WriteComplexImageAsCSV("qi-p1.txt", (std::complex<float>*) &p1[0], 2*qi_FFT_length, njobs);

	for (int j=0;j<njobs;j++) {
		float2* r1 = &p1[j * cfg.qi_radialsteps * 4];
		float2* r0 = &p0[j * cfg.qi_radialsteps * 4];

		float* s1 = &q1[j * cfg.qi_radialsteps * 4];
		float* s0 = &q0[j * cfg.qi_radialsteps * 4];

		for (int q=0;q<4;q++) {
			for (int r=0;r<cfg.qi_radialsteps;r++) {

				s1 ++;
				s0 ++;
			}
		}


	}
	*/
	checksum(s->d_quadrants.data, qi_FFT_length * 2, njobs, "quadrant");
	checksum(s->d_QIprofiles.data, qi_FFT_length * 2, njobs, "prof");
	checksum(s->d_QIprofiles_reverse.data, qi_FFT_length * 2, njobs, "revprof");

	hipfftComplex* prof = (hipfftComplex*)s->d_QIprofiles.data;
	hipfftComplex* revprof = (hipfftComplex*)s->d_QIprofiles_reverse.data;

	hipfftExecC2C(s->fftPlan, prof, prof, HIPFFT_FORWARD);
	hipfftExecC2C(s->fftPlan, revprof, revprof, HIPFFT_FORWARD);

	int nval = qi_FFT_length * 2 * batchSize, nthread=256;
	QI_MultiplyWithConjugate<<< dim3( (nval + nthread - 1)/nthread ), dim3(nthread), 0, s->stream >>>(nval, prof, revprof);
	hipfftExecC2C(s->fftPlan, prof, prof, HIPFFT_BACKWARD);

	float2* d_offsets=0;
	float pixelsPerProfLen = (cfg.qi_maxradius-cfg.qi_minradius)/cfg.qi_radialsteps;
	QI_OffsetPositions<<<blocks(njobs), threads(), 0, s->stream>>>
		(njobs, initial->data, newpos->data, prof, qi_FFT_length, d_offsets, pixelsPerProfLen, s->d_shiftbuffer.data); 
}


template<typename TImageSampler>
void QueuedCUDATracker::ExecuteBatch(Stream *s)
{
	if (s->JobCount()==0)
		return;
	//dbgprintf("Sending %d images to GPU stream %p...\n", s->jobCount, s->stream);

	Device *d = s->device;
	hipSetDevice(d->index);
	kernelParams.qi.radialgrid = d->d_qiradialgrid.data;
	kernelParams.zlut.img = d->zlut;
	kernelParams.zlut.radialgrid = d->d_zlutradialgrid.data;
	kernelParams.zlut.zcmpwindow = d->zcompareWindow.data;

	hipEventRecord(s->batchStart, s->stream);

	
	{ProfileBlock p("image to gpu");
	s->images.copyToDevice(s->hostImageBuf.data(), true, s->stream); }
	//hipMemcpy2DAsync( s->images.data, s->images.pitch, s->hostImageBuf.data(), sizeof(float)*s->images.w, s->images.w*sizeof(float), s->images.h * s->JobCount(), hipMemcpyHostToDevice, s->stream); }
	//{ ProfileBlock p("jobs to gpu");
	//s->d_jobs.copyToDevice(s->jobs.data(), s->jobCount, true, s->stream); }
	hipEventRecord(s->imageCopyDone, s->stream);

	TImageSampler::BindTexture(s->images);
	{ ProfileBlock p("COM");
	BgCorrectedCOM<TImageSampler> <<< blocks(s->JobCount()), threads(), 0, s->stream >>> 
		(s->JobCount(), s->images, s->d_com.data, s->d_imgmeans.data, cfg.com_bgcorrection);
	checksum(s->d_com.data, 1, s->JobCount(), "com");
	}
	hipEventRecord(s->comDone, s->stream);

//	{ ProfileBlock p("COM results to host");
	s->d_com.copyToHost(s->com.data(), true, s->stream);

	device_vec<float3> *curpos = &s->d_com;
	if (s->localizeFlags & LocalizeQI) {
		ProfileBlock p("QI");
		for (int a=0;a<cfg.qi_iterations;a++) {
			QI_Iterate<TImageSampler> (curpos, &s->d_resultpos, s);
			curpos = &s->d_resultpos;
		}
	}
	hipEventRecord(s->qiDone, s->stream);

	// Compute radial profiles
	if (s->localizeFlags & (LocalizeZ | LocalizeBuildZLUT)) {
		dim3 numThreads(16, 16);
		dim3 numBlocks( (s->JobCount() + numThreads.x - 1) / numThreads.x, (cfg.zlut_radialsteps + numThreads.y - 1) / numThreads.y);
		{ ProfileBlock p("ZLUT radial profile");
		ZLUT_RadialProfileKernel<TImageSampler> <<< numBlocks , numThreads, 0, s->stream >>>
			(s->JobCount(), s->images, kernelParams.zlut, curpos->data, s->d_radialprofiles.data,  s->d_imgmeans.data); }

		{ ProfileBlock p("ZLUT normalize profiles");
		ZLUT_NormalizeProfiles<<< blocks(s->JobCount()), threads(), 0, s->stream >>> (s->JobCount(), kernelParams.zlut, s->d_radialprofiles.data); }

		s->d_zlutmapping.copyToDevice(s->zlutmapping.data(), s->JobCount(), true, s->stream);
	}
	// Store profile in LUT
	if (s->localizeFlags & LocalizeBuildZLUT) {
		{ ProfileBlock p("ZLUT build zlut");
		ZLUT_ProfilesToZLUT <<< blocks(s->JobCount()), threads(), 0, s->stream >>> (s->JobCount(), s->images, kernelParams.zlut, curpos->data, s->d_zlutmapping.data, s->d_radialprofiles.data); }
	}
	// Compute Z 
	if (s->localizeFlags & LocalizeZ) {
		int zplanes = kernelParams.zlut.planes;
		dim3 numThreads(8, 16);
		{ ProfileBlock p("ZLUT compute Z");
		ZLUT_ComputeProfileMatchScores <<< dim3( (s->JobCount() + numThreads.x - 1) / numThreads.x, (zplanes  + numThreads.y - 1) / numThreads.y), numThreads, 0, s->stream >>> 
			(s->JobCount(), kernelParams.zlut, s->d_radialprofiles.data, s->d_zlutcmpscores.data, s->d_zlutmapping.data);
		ZLUT_ComputeZ <<< blocks(s->JobCount()), threads(), 0, s->stream >>> (s->JobCount(), kernelParams.zlut, curpos->data, s->d_zlutcmpscores.data, s->d_zlutmapping.data);
		}
	}
	TImageSampler::UnbindTexture(s->images);
	hipEventRecord(s->zcomputeDone, s->stream);

	{ ProfileBlock p("Results to host");
	curpos->copyToHost(s->results.data(), true, s->stream);}

	// Make sure we can query the all done signal
	hipEventRecord(s->localizationDone, s->stream);

	s->state = Stream::StreamExecuting;
}


int QueuedCUDATracker::FetchResults()
{
	for (uint a=0;a<streams.size();a++) {
		Stream* s = streams[a];
		if (s->state == Stream::StreamExecuting && s->IsExecutionDone()) {
			s->lock();
			CopyStreamResults(s);
			s->state = Stream::StreamIdle;
			s->unlock();
		}
	}
	return results.size();
}

void QueuedCUDATracker::CopyStreamResults(Stream *s)
{
	for (int a=0;a<s->JobCount();a++) {
		LocalizationJob& j = s->jobs[a];

		LocalizationResult r;
		r.job = j;
		r.firstGuess =  vector2f( s->com[a].x, s->com[a].y );
		r.pos = vector3f( s->results[a].x , s->results[a].y, s->results[a].z);
		if ( !(s->jobs[a].locType & LocalizeZ))
			r.pos.z = 0.0f;

		results.push_back(r);
	}

	// Update times
	float qi, com, imagecopy, zcomp;
	hipEventElapsedTime(&imagecopy, s->batchStart, s->imageCopyDone);
	hipEventElapsedTime(&com, s->imageCopyDone, s->comDone);
	hipEventElapsedTime(&qi, s->comDone, s->qiDone);
	hipEventElapsedTime(&zcomp, s->qiDone, s->zcomputeDone);
	time_COM += com;
	time_QI += qi;
	time_imageCopy += imagecopy;
	time_ZCompute += zcomp;
	batchesDone ++;
	
	s->jobs.clear();
	s->localizeFlags = 0; // reset this for the next batch
}

int QueuedCUDATracker::PollFinished(LocalizationResult* dstResults, int maxResults)
{
	FetchResults();

	int numResults = 0;
	while (numResults < maxResults && !results.empty()) {
		dstResults[numResults++] = results.front();
		results.pop_front();
	}
	return numResults;
}

// data can be zero to allocate ZLUT data
void QueuedCUDATracker::SetZLUT(float* data,  int numLUTs, int planes, float* zcmp) 
{
	kernelParams.zlut.planes = planes;
	
	for (uint i=0;i<devices.size();i++) {
		devices[i]->SetZLUT(data, cfg.zlut_radialsteps, planes, numLUTs, zcmp);
	}

	for (uint i=0;i<streams.size();i++) {
		StreamUpdateZLUTSize(streams[i]);
	}
}

void QueuedCUDATracker::StreamUpdateZLUTSize(Stream* s)
{		
	hipSetDevice(s->device->index);
	s->d_zlutcmpscores.init(s->device->zlut.h * batchSize);
}

void QueuedCUDATracker::Device::SetZLUT(float *data, int radialsteps, int planes, int numLUTs, float* zcmp)
{
	hipSetDevice(index);

	if (zcmp)
		zcompareWindow.copyToDevice(zcmp, radialsteps, false);
	else 
		zcompareWindow.free();

	zlut = cudaImageListf::alloc(radialsteps, planes, numLUTs);
	if (data) {
		for (int i=0;i<numLUTs;i++)
			zlut.copyImageToDevice(i, &data[planes*radialsteps*i]);
	}
	else zlut.clear();
}	

// delete[] memory afterwards
float* QueuedCUDATracker::GetZLUT(int *count, int* planes)
{
	cudaImageListf* zlut = &devices[0]->zlut;

	float* data = new float[zlut->h * cfg.zlut_radialsteps * zlut->count];
	if (zlut->data) {
		//zlut->copyToHost(data, false);
		for (int i=0;i<zlut->count;i++)
			zlut->copyImageToHost(i, &data[cfg.zlut_radialsteps*zlut->h]);
	} else
		std::fill(data, data+(cfg.zlut_radialsteps*zlut->h*zlut->count), 0.0f);

	if (planes) *planes = zlut->h;
	if (count) *count = zlut->count;

	return data;
}


int QueuedCUDATracker::GetResultCount()
{
	return FetchResults();
}


void QueuedCUDATracker::ScheduleFrame(uchar *imgptr, int pitch, int width, int height, ROIPosition *positions, int numROI, QTRK_PixelDataType pdt, const LocalizationJob* jobInfo)
{
	uchar* img = (uchar*)imgptr;
	int bpp = sizeof(float);
	if (pdt == QTrkU8) bpp = 1;
	else if (pdt == QTrkU16) bpp = 2;
	for (int i=0;i<numROI;i++){
		ROIPosition pos = positions[i];
		if (pos.x < 0 || pos.y < 0 || pos.x + cfg.width > width || pos.y + cfg.height > height)
			continue;

		uchar *roiptr = &img[pitch * pos.y + pos.x * bpp];
		LocalizationJob job = *jobInfo;
		job.zlutIndex = i + jobInfo->zlutIndex;
		ScheduleLocalization(roiptr, pitch, pdt, &job);
	}
}

std::string QueuedCUDATracker::GetProfileReport()
{
	float f = 1.0f/batchesDone;

	return "CUDA tracker report: " + SPrintf("%d batches done of size %d", batchesDone, batchSize ) + "\n" +
		SPrintf("Image copying: %f ms per image\n", time_imageCopy*f) +
		SPrintf("QI: %f ms per image\n", time_QI*f) +
		SPrintf("COM: %f ms per image\n", time_COM*f) +
		SPrintf("Z Computing: %f ms per image\n", time_ZCompute*f);
}


