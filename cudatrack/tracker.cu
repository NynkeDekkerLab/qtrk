#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "std_incl.h"

#include <stdio.h>

#include "tracker.h"
#include "Array2D.h"
#include "utils.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define LSQ_FDECL __host__ __device__
#include "LsqQuadraticFit.h"

using namespace gpuArray;

void throwCudaError(hipError_t err)
{
	std::string msg = SPrintf("CUDA error: %s", hipGetErrorString(err));
	dbgout(msg);
	throw std::runtime_error(msg);
}


template<typename T>
void safeCudaFree(T*& ptr) {
	if (ptr) {
		hipFree(ptr);
		ptr = 0;
	}
}

class TrackerBuffer
{
public:
	Array2D<pixel_t, float>* image;
	reducer_buffer<float> reduceBuffer;
	thrust::device_vector<pixel_t> sortBuf;
	pixel_t* h_image;

	TrackerBuffer(uint w,uint h) : reduceBuffer(w,h) {
		image = new Array2D<pixel_t,float>(w, h);
		h_image = new pixel_t[w*h];
	}
	~TrackerBuffer()
	{
		if (h_image) delete[] h_image;
		if (image) delete image;
	}
};

Tracker::Tracker(uint w, uint h) {
	magic = TRACKER_MAGIC;

	width = w;
	height = h;
	buffer = new TrackerBuffer(w,h);
}

Tracker::~Tracker() {
	delete buffer;
}

void Tracker::setImage(pixel_t* data, uint pitchInBytes) {
	buffer->image->set(data, pitchInBytes);
}


struct TestImgComputePixel {
	float xpos, ypos, S;
	float compute(uint x, uint y) {
/*		if (x==0&&y==0)
			printf("value: %f", value);
*/
		float X = x + 0.5f - xpos;
		float Y = y + 0.5f - ypos;
		float r = sqrtf(X*X+Y*Y)+1;
		float v = sinf( (r-10)*2*3.141593f*S);
		return v*v / (r * r * S);
	}
};


void Tracker::loadTestImage(float xpos, float ypos, float S)
{
	TestImgComputePixel pixel_op = { xpos, ypos, 1.0f/S };

	// generate
	float* buf = new float[width*height];
	for (uint y=0;y<height;y++)
		for(uint x=0;x<width;x++)
			buf[y*width+x] = pixel_op.compute(x,y);

	// normalize
	float minv, maxv;
	minv=maxv=buf[0];
	for (int k=0;k<width*height;k++) {
		minv=std::min(minv, buf[k]);
		maxv=std::max(maxv, buf[k]);
	}
	// convert to uchar
	uchar *ibuf = new uchar[width*height];
	for (int k=0;k<width*height;k++)
		ibuf[k]= 255.0f * (buf[k]-minv)/(maxv-minv);
	delete[] buf;

	buffer->image->set(ibuf, sizeof(pixel_t)*width);
	memcpy(buffer->h_image, ibuf, sizeof(pixel_t)*width*height);
	delete[] ibuf;
}

vector2f Tracker::computeCOM()
{
	if (!buffer->image)
		return vector2f();

	vector2f com;
	com.x = buffer->image->momentX(buffer->reduceBuffer);
	com.y = buffer->image->momentY(buffer->reduceBuffer);
	float sum = buffer->image->sum(buffer->reduceBuffer);
	com.x /= sum;
	com.y /= sum;
	return com;
}


vector2f Tracker::computeBgCorrectedCOM()
{
	if (!buffer->image)
		return vector2f();

	pixel_t median = computeMedianPixelValue();

	vector2f com;
	com.x = buffer->image->momentX(buffer->reduceBuffer);
	com.y = buffer->image->momentY(buffer->reduceBuffer);
	float sum = buffer->image->sum(buffer->reduceBuffer);
	com.x /= sum;
	com.y /= sum;
	return com;
}

vector2f Tracker::XCorLocalize(vector2f initial)
{
	vector2f estimate;

	// bind the image as texture
	texture<pixel_t, hipTextureType1D, hipReadModeNormalizedFloat> tex;

	buffer->image->bindTexture(tex);

	buffer->image->unbindTexture(tex);

	return initial;
}

void Tracker::copyToHost(pixel_t* data, uint pitchInBytes)
{
	if (buffer->image)
		buffer->image->copyToHost(data, pitchInBytes);
}

void* Tracker::getCurrentBufferImage() {
	return buffer->image;
}

pixel_t Tracker::computeMedianPixelValue() {
	buffer->image->copyTo(buffer->sortBuf);
	thrust::sort(buffer->sortBuf.begin(), buffer->sortBuf.end());
	pixel_t median = buffer->sortBuf[buffer->sortBuf.size()/2];
	return median;
}
