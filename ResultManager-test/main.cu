#include "hip/hip_runtime.h"
#include "std_incl.h"
#include "hip/hip_runtime.h"
#include "" 
#include <stdio.h>
#include <iostream>
#include <string>
#include <ctime> 
#include "omp.h"
#include <stdio.h>
#include <fstream>
#include <sys/stat.h>
#include <vector>
#include <boost/tokenizer.hpp>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_01.hpp> 
#include <boost/filesystem/operations.hpp> 
#include <boost/chrono.hpp> 
#include <memory>
#include "../cputrack/ResultManager.h"
#include "../cputrack/QueuedTracker.h" 
#include "../cputrack/QueuedCPUTracker.h" 
#include "../cudatrack/QueuedCUDATracker.h" 
#include "../cputrack-test/SharedTests.h"
/*
This file tests the Result Manager. It is pretty straightforward:
- Build the Result Manager as LabView would
- Overload it with Data
- Test whether it loses frames during overload



*/
int main(int argc, char* argv[])
{
	auto timeStart = boost::chrono::high_resolution_clock::now();
	fprintf(stderr, "Note: Initialising ndlab/test/ResultManager (%d arguments).\n", argc);

	if (argc != 3)
	{
		fprintf(stderr, "You have to give in N, the number of images (multiple of 4) and \n\t gpu or cpu.\n");
		return 0;
	}
	//number of images/frames
	const int N = (int)atoi(argv[1]); //4 beads, 24 images; 6 frames per image?

	fprintf(stderr, "Testing ResultManager with %d images. %s \n", N, argv[1]);

	const char* file = "./ResultManagerData.txt";
	const char * frameinfo = "./ResultManagerFrameInfo.txt";
	//shared_ptrs are far superior in terms of memory management etc.
	// make_shared is exception safe.
	std::shared_ptr<ResultManagerConfig> cfg = std::make_shared<ResultManagerConfig>();
	cfg->numBeads = 4;
	cfg->numFrameInfoColumns = 0;
	cfg->scaling = vector3f(1.0, 1.0, 1.0);
	cfg->offset = vector3f(0.0, 0.0, 0.0);
	cfg->writeInterval = 2500;
	cfg->maxFramesInMemory = 0;// 100000;
	cfg->binaryOutput = false;

	std::vector< std::string > colNames;
	std::string testName("Hey now");
	colNames.push_back(testName);

	fprintf(stderr, "Allocating Result Manager now. \n");
	// The GodFather manages your results.
	std::shared_ptr<ResultManager> manager = std::make_shared<ResultManager>(file, frameinfo, cfg.get(), colNames);
	//The QueuedCPUTracker instance is required to retrieve the results. It needs settings.
	QTrkComputedConfig settings;
	settings.qi_iterations = 2;
	settings.zlut_minradius = 1;
	settings.qi_minradius = 1;
	settings.width = settings.height = 100;
	settings.Update();

	std::string fileName = "./exp.jpg";

	bool fileExists = boost::filesystem::exists(fileName);

	if (!fileExists)
	{
		fprintf(stderr, "File %s not found; is it in the directory of the executable?\n\n", fileName.c_str());
		return 0;
	}

	//Let's load some image data.
	auto data = ReadJPEGFile(fileName.c_str());

	std::shared_ptr<QueuedTracker> qtrk;

	if (argc == 3)
	{
		std::string argTracker = std::string(argv[2]);
		if (argTracker == "gpu")
		{
			fprintf(stderr, "Using CUDA tracker (GPU).\n");
			std::shared_ptr<QueuedCUDATracker> cudaTracker = std::make_shared<QueuedCUDATracker>(settings);
			cudaTracker->EnableTextureCache(true);

			qtrk = cudaTracker;
		}
		else if(argTracker == "cpu")
		{
			fprintf(stderr, "Using CPU tracker (CPU).\n");
			qtrk = std::make_shared<QueuedCPUTracker>(settings);
		}
		else
		{
			fprintf(stderr, "No tracker specified. Choose either cpu or gpu.\n");
			return 0;
		}
	}
	else
	{
		fprintf(stderr, "Faulty arguments. Your mother was a hamster, %d th of her name.", argc);
		return 0;
	}
	//localization Mode QI tracker
	auto modeQI = (LocMode_t)(LT_QI | LT_NormalizeProfile | LT_LocalizeZ);

	qtrk->SetLocalizationMode(modeQI);


	std::vector<LocalizationJob> jobs;

	int frame = 0;
	for (int i = 0; i < N; i++)
	{
		if (i % 4 == 0 && i != 0)
		{
			frame++;
		}
		//Make a localization job (batch of calculations)
		LocalizationJob job(frame, 0, 0, 0);
		job.zlutIndex = i % 4;
		jobs.push_back(job);
		qtrk->ScheduleImageData(&data, &job);
	}
	manager->SetTracker(qtrk.get());
	//Process images (using Flush because Start is CPU only)
	qtrk->Flush();
	int i = 0;
	while (manager->GetFrameCounters().localizationsDone < N)
	{
		if (i > 100000)
		{
			auto counters = manager->GetFrameCounters();
			fprintf(stderr, "Update: %d Localisations performed.\n", counters.localizationsDone);
			i = 0;
		}
		i++;
	}

	//Assign tracker

	//Assign frame
	float somefloat = 0.0;
	for (int i = 0; i < N / 4; i++)
	{
		manager->StoreFrameInfo(i, i, &somefloat);
	}
	//Pointer that will be filled with results
	std::vector<LocalizationResult> results;

	vector3f startPosition(0.0f, 0.0f, 0.0f);
	vector2f initialGuess(45.0f, 50.0f);

	for (int i = 0; i < N; i++)
	{
		LocalizationResult currentResult;
		currentResult.job = jobs.at(i);
		currentResult.pos = startPosition;
		currentResult.firstGuess = initialGuess;
		currentResult.error = 0;
		currentResult.imageMean = 0.0f;
		results.push_back(currentResult);
	}

	//Fill results array
	manager->Flush();
	//Wait untill all localizations have been performed.

	i = 0;
	while (manager->GetFrameCounters().lastSaveFrame  < N / 4)
	{
		if (i > 100000)
		{
			auto counters = manager->GetFrameCounters();

			fprintf(stderr, "Update[%.3f]: %d frames saved.\n", i, counters.lastSaveFrame);
		}
	}

	auto counters = manager->GetFrameCounters();
	auto getResults = manager->GetResults(results.data(), 0, N / cfg->numBeads);
	fprintf(stderr, "ResultManager results (%d) :\n", getResults);
	fprintf(stderr, "\t frame\t bead\t x\t y\t z\n");
	for (unsigned int i = (results.size() - 25)>0 ? (results.size() - 25) : 0; i < results.size(); i++)
	{
		auto result = results[i];
		fprintf(stderr, "\t%d\t%d\t%.3f\t%.3f\t%.3f\n", result.job.frame, i % 4, result.pos.x, result.pos.y, result.pos.z);

	}
	//Report final information

	printf("Frame counters:\n\t Started at %d, processed %d, finished on %d\n", counters.startFrame, counters.processedFrames, counters.lastSaveFrame);
	printf("\tCaptured %d, localizations %d, lostFrames %d, file error %d.\n", counters.capturedFrames, counters.localizationsDone, counters.lostFrames, counters.fileError);
 
    //report time, end program 
	auto timeEnd = boost::chrono::high_resolution_clock::now();
	auto microSeconds = boost::chrono::duration_cast<boost::chrono::microseconds>(timeEnd - timeStart).count();
	fprintf(stderr, "Note: Elapsed time %ld microseconds. \n", microSeconds);
	return 0;
}

