#include "hip/hip_runtime.h"
#include "std_incl.h"
#include "hip/hip_runtime.h"
#include "" 
#include <stdio.h>
#include <iostream>
#include <string>
#include <ctime> 
#include "omp.h"
#include <stdio.h>
#include <fstream>
#include <sys/stat.h>
#include <vector>
#include <boost/tokenizer.hpp>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_01.hpp> 
#include <boost/filesystem/operations.hpp> 
#include <boost/chrono.hpp> 
#include <memory>
#include "../cputrack/ResultManager.h"
#include "../cputrack/QueuedTracker.h" 
#include "../cputrack/QueuedCPUTracker.h" 
#include "../cudatrack/QueuedCUDATracker.h" 
#include "../cputrack-test/SharedTests.h"
/*
This file tests the Result Manager. It is pretty straightforward:
- Build the Result Manager as LabView would
- Overload it with Data
- Test whether it loses frames during overload



*/
int main(int argc, char* argv[])
{
	//At the end, we report the time taken.
	auto timeStart = boost::chrono::high_resolution_clock::now();
	//Just a reminder.
	fprintf(stderr, "Note: Initialising ndlab/test/ResultManager (%d arguments).\n", argc);
	//Otherwise, it will throw errors.
	if (argc != 3)
	{
		fprintf(stderr, "You have to give in N, the number of images (multiple of 4) and \n\t gpu or cpu.\n");
		return 0;
	}
	//Number of images, which are inputted as if there are 4 beads (which really doesn't matter).
	const int N = (int)atoi(argv[1]); //4 beads, 24 images; 6 frames per image?
	//Always report back.
	fprintf(stderr, "Testing ResultManager with %d images. %s \n", N, argv[1]);
	//Where to safe data/frameinfo. 
	const char* file = "./ResultManagerData.txt";
	const char * frameinfo = "./ResultManagerFrameInfo.txt";
	// Better to use modern smart pointers. Config settings are largely taken from older tests.
	std::shared_ptr<ResultManagerConfig> cfg = std::make_shared<ResultManagerConfig>();
	cfg->numBeads = 4;
	cfg->numFrameInfoColumns = 0;
	cfg->scaling = vector3f(1.0f, 1.0f, 1.0f);
	cfg->offset = vector3f(0.0f, 0.0f, 0.0f);
	cfg->writeInterval = 2500;
	cfg->maxFramesInMemory = 0;// 100000;
	cfg->binaryOutput = false;

	std::vector< std::string > colNames;
	std::string testName("Hey now");
	colNames.push_back(testName);

	fprintf(stderr, "Allocating Result Manager now. \n");
	// The manager is an instance of the Result Manager class. Surprise.
	std::shared_ptr<ResultManager> manager = std::make_shared<ResultManager>(file, frameinfo, cfg.get(), colNames);
	//The QueuedCPUTracker instance is required to retrieve the results. It needs settings.
	QTrkComputedConfig settings;
	settings.qi_iterations = 5;
	settings.zlut_minradius = 1;
	settings.qi_minradius = 1;
	settings.width = settings.height = 100;
	settings.Update();
	//Experimental file to use for test.
	std::string fileName = "./exp.jpg";
	//Make sure the file exists; otherwise, it will throw a really weird error.
	bool fileExists = boost::filesystem::exists(fileName);

	if (!fileExists)
	{
		fprintf(stderr, "File %s not found; is it in the directory of the executable?\n\n", fileName.c_str());
		return 0;
	}

	//Let's load some image data.
	auto data = ReadJPEGFile(fileName.c_str());

	std::shared_ptr<QueuedTracker> qtrk;
	//Make sure we are using the right tracker (as requested)
	if (argc == 3)
	{
		std::string argTracker = std::string(argv[2]);
		if (argTracker == "gpu")
		{
			fprintf(stderr, "Using CUDA tracker (GPU).\n");
			std::shared_ptr<QueuedCUDATracker> cudaTracker = std::make_shared<QueuedCUDATracker>(settings);
			cudaTracker->EnableTextureCache(true);

			qtrk = cudaTracker;
		}
		else if(argTracker == "cpu")
		{
			fprintf(stderr, "Using CPU tracker (CPU).\n");
			qtrk = std::make_shared<QueuedCPUTracker>(settings);
		}
		else
		{
			fprintf(stderr, "No tracker specified. Choose either cpu or gpu.\n");
			return 0;
		}
	}
	else
	{
		fprintf(stderr, "Faulty arguments. Your mother was a hamster, %d th of her name.", argc);
		return 0;
	}
	//localization Mode QI tracker
	auto modeQI = (LocMode_t)(LT_QI | LT_NormalizeProfile | LT_LocalizeZ);

	qtrk->SetLocalizationMode(modeQI);

	//Make the calculations
	std::vector<LocalizationJob> jobs;

	int frame = 0;
	for (int i = 0; i < N; i++)
	{
		if (i % 4 == 0 && i != 0)
		{
			frame++;
		}
		//Make a localization job (batch of calculations)
		LocalizationJob job(frame, 0, 0, 0);
		job.zlutIndex = i % 4; //actually, frame number
		jobs.push_back(job);
		qtrk->ScheduleImageData(&data, &job);
	}
	manager->SetTracker(qtrk.get());
	//Process images (using Flush because Start is CPU only)
	qtrk->Flush();
	//Report progress
	int i = 0;
	while (manager->GetFrameCounters().localizationsDone < N)
	{
		if (i > 100000)
		{
			auto counters = manager->GetFrameCounters();
			fprintf(stderr, "Update: %d Localisations performed.\n", counters.localizationsDone);
			i = 0;
		}
		i++;
	}
	 
	//Assign frame
	float somefloat = 0.0;
	for (int i = 0; i < N / 4; i++)
	{
		manager->StoreFrameInfo(i, i, &somefloat);
	}
	//Pointer that will be filled with results
	std::vector<LocalizationResult> results;

	vector3f startPosition(0.0f, 0.0f, 0.0f);
	vector2f initialGuess(45.0f, 50.0f);
	//Allocate the 'array'  that will hold the results.
	for (int i = 0; i < N; i++)
	{
		LocalizationResult currentResult;
		currentResult.job = jobs.at(i);
		currentResult.pos = startPosition;
		currentResult.firstGuess = initialGuess;
		currentResult.error = 0;
		currentResult.imageMean = 0.0f;
		results.push_back(currentResult);
	}

	//Fill results array
	manager->Flush();
	//Wait untill all localizations have been performed.

	i = 0;
	while (manager->GetFrameCounters().lastSaveFrame  < N / 4)
	{
		if (i > 100000)
		{
			auto counters = manager->GetFrameCounters();

			fprintf(stderr, "Update[%.3f]: %d frames saved.\n", i, counters.lastSaveFrame);
		}
	}
	//Get the results
	auto counters = manager->GetFrameCounters();
	auto getResults = manager->GetResults(results.data(), 0, N / cfg->numBeads);
	//Report results
	fprintf(stderr, "ResultManager results (%d) :\n", getResults);
	fprintf(stderr, "\t frame\t bead\t x\t y\t z\n");
	for (unsigned int i = (results.size() - 25)>0 ? (results.size() - 25) : 0; i < results.size(); i++)
	{
		auto result = results[i];
		fprintf(stderr, "\t%d\t%d\t%.3f\t%.3f\t%.3f\n", result.job.frame, i % 4, result.pos.x, result.pos.y, result.pos.z);

	}
	//Report final information

	printf("Frame counters:\n\t Started at %d, processed %d, finished on %d\n", counters.startFrame, counters.processedFrames, counters.lastSaveFrame);
	printf("\tCaptured %d, localizations %d, lostFrames %d, file error %d.\n", counters.capturedFrames, counters.localizationsDone, counters.lostFrames, counters.fileError);
 
    //report time, end program 
	auto timeEnd = boost::chrono::high_resolution_clock::now();
	auto microSeconds = boost::chrono::duration_cast<boost::chrono::microseconds>(timeEnd - timeStart).count();
	fprintf(stderr, "Note: Elapsed time %ld microseconds. \n", microSeconds);
	return 0;
}

